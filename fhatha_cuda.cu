
#include <hip/hip_runtime.h>
#ifdef _ENABLE_CUDA

#include "fhatha.h"
#include <cuda.h>
//#include <cutil.h>
//#include <cutil_inline.h>

#include "cuda_extmath.h"

__global__ void cuda_fhatha_kernel(float_type* data,  size_t N, float_type* phi_buf, float_type* phi_mult, float_type* jbuf, float_type* x, size_t istride, size_t cstride, size_t dist, size_t pointN);

__global__ void cuda_qdht_kernel(float_type* data, float_type* buf, size_t N, float_type* C, float_type* m1,  size_t istride, size_t cstride, size_t idist, size_t pointN);

void fhatha_runmany_cuda(fhatha_plan* p, f_complex* data, size_t nN, size_t stride, size_t dist, bool do_extra_transfer, float_type* cuda_buf)
{
  float_type* cuda_data;
  float_type* cuda_phibuf; 
  float_type* cuda_phimult;
  float_type* cuda_j1buf; 
  float_type* cuda_x;
  size_t device_freemem=0, device_totalmem=0;
 
  (cudaMemGetInfo(&device_freemem, &device_totalmem));

  size_t cuda_pointN = exp2(floor(log2((double)device_freemem/p->N/sizeof(f_complex)/3)));
  if (cuda_pointN > nN) cuda_pointN=nN;

  int    blocksize = 64;  if (blocksize > cuda_pointN) blocksize=cuda_pointN;
  size_t cuda_piece_size_ = p->N*cuda_pointN*sizeof(f_complex);

  if (do_extra_transfer) 
  {
   (cudaMalloc((void**)&cuda_data,          cuda_piece_size_));
   (cudaMalloc((void**)&cuda_phibuf,      2*cuda_piece_size_));
  }
  

  (cudaMalloc((void**)&cuda_j1buf,       2*(p->N)*sizeof(f_complex ))); 
  (cudaMalloc((void**)&cuda_phimult,       (p->N)*sizeof(float_type)));
  (cudaMalloc((void**)&cuda_x,             (p->N)*sizeof(float_type)));

  (cudaMemcpy(cuda_j1buf,   p->j1,      2*sizeof(f_complex) *(p->N), cudaMemcpyHostToDevice));
  (cudaMemcpy(cuda_phimult, p->phi_mult,  sizeof(float_type)*(p->N), cudaMemcpyHostToDevice));
  (cudaMemcpy(cuda_x,       p->x,         sizeof(float_type)*(p->N), cudaMemcpyHostToDevice));

  if (do_extra_transfer)
  {
  if (cuda_pointN < nN)
  {
   float_type* data_restrided = (float_type*)malloc_ch(cuda_piece_size_); 
   int newstride = cuda_pointN;
 
   for (long i=0; i<nN; i+= cuda_pointN)
   { 
    for (int ni=0; ni<cuda_pointN; ni++)
    for (int nt=0; nt<p->N; nt++) 
    {
	   data_restrided[ni +  2*nt   *newstride]  = real(data[nt*stride + (i+ni)*dist]);
	   data_restrided[ni + (2*nt+1)*newstride]  = imag(data[nt*stride + (i+ni)*dist]);	

	   //data_restrided[ni%(newstride)+(2*nt   + 2*ni/newstride*p->N)*newstride] = real(data[nt*stride + i*dist]);
	   //data_restrided[ni%(newstride)+(2*nt+1 + 2*ni/newstride*p->N)*newstride] = imag(data[nt*stride + i*dist]);
    }
    (cudaMemcpy(cuda_data, data_restrided, cuda_piece_size_, cudaMemcpyHostToDevice)) ;
    cuda_fhatha_kernel<<< cuda_pointN/blocksize,blocksize >>>(cuda_data, p->N, cuda_phibuf, cuda_phimult, cuda_j1buf, cuda_x, 2*newstride, newstride, 1, cuda_pointN);
    (cudaMemcpy(data_restrided, cuda_data, cuda_piece_size_, cudaMemcpyDeviceToHost));
   
    for (int ni=0; ni<cuda_pointN; ni++)
    for (int nt=0; nt<p->N; nt++) 
    {
	   //data[nt*stride+ni*dist] = f_complex(data_restrided[ni%newstride+(2*nt +   ni/newstride*p->N)*newstride], \
		                                   data_restrided[ni%newstride+(2*nt+1 + ni/newstride*p->N)*newstride]);
	   data[nt*stride+(i+ni)*dist] = f_complex(data_restrided[ni +  2*nt   *newstride],\
		                                       data_restrided[ni + (2*nt+1)*newstride]); 
    }
   }
   free(data_restrided);
  }
  else
  {
	  (cudaMemcpy(cuda_data, data, cuda_piece_size_, cudaMemcpyHostToDevice));
	  cuda_fhatha_kernel<<<cuda_pointN/blocksize, blocksize>>>(cuda_data, p->N, cuda_phibuf, cuda_phimult, cuda_j1buf, cuda_x, 2*stride, 1, 2*dist, cuda_pointN);
	  (cudaMemcpy(data, cuda_data, cuda_piece_size_, cudaMemcpyDeviceToHost));
  }
  

  
  (cudaFree(cuda_phibuf));
  (cudaFree(cuda_data));
  }
  else
  {
        cuda_fhatha_kernel<<<cuda_pointN/blocksize, blocksize>>>((float_type*)data, p->N, cuda_buf, cuda_phimult, cuda_j1buf, cuda_x, 2*stride, 1, 2*dist, cuda_pointN);
  }

  (cudaFree(cuda_j1buf));
  (cudaFree(cuda_phimult));
  (cudaFree(cuda_x));
}


__global__ void cuda_fhatha_kernel(float_type* data, size_t N, float_type* phi_buf, float_type* phi_mult, float_type* jbuf, float_type* x, size_t istride, size_t cstride, size_t idist, size_t pointN)
{
	size_t pi      = blockDim.x*blockIdx.x + threadIdx.x; 
	size_t hstride = pointN; 

	data += pi*idist;
	phi_buf += pi;

	for (size_t nt=0; nt < (N-1); nt++) 
	{
		float_type phi_mult_ = phi_mult[nt]; 
		phi_buf[(2*nt)  *hstride] = phi_mult_*(data[(nt)*istride]          -data[(nt+1)*istride        ]); 
		phi_buf[(2*nt+1)*hstride] = phi_mult_*(data[(nt)*istride+cstride]  -data[(nt+1)*istride+cstride]); 
	}

	phi_buf[(2*N-2)*hstride] = data[(N-1)*istride];
	phi_buf[(2*N-1)*hstride] = data[(N-1)*istride+cstride];

	for (size_t nt=2*N; nt < 4*N; nt++) 
	{
		phi_buf[ nt   *hstride] = 0;
	}

	fft_device_strided(phi_buf, 2*N, -1, hstride);

	for (size_t nt=0; nt<2*N; nt++)
	{
		float_type phi_buf_re = phi_buf[(2*nt  )*hstride];
		float_type phi_buf_im = phi_buf[(2*nt+1)*hstride];
		float_type j1re       = jbuf[2*nt];
		float_type j1im       = jbuf[2*nt+1];
		
		phi_buf[(2*nt)  *hstride] = phi_buf_re*j1re - phi_buf_im*j1im;
		phi_buf[(2*nt+1)*hstride] = phi_buf_im*j1re + phi_buf_re*j1im;
	}
	
	fft_device_strided(phi_buf, 2*N, -1, hstride); 
		
	for (size_t nt=0; nt<N; nt++) 
	{
		data[nt*istride]         = phi_buf[ 2*nt   *hstride]/x[nt];
		data[nt*istride+cstride] = phi_buf[(2*nt+1)*hstride]/x[nt];
	}
}


void qdht_runmany_cuda(qdht_plan* p, f_complex* data, size_t nN, size_t stride, size_t dist, bool do_extra_transfer, float_type* cuda_buf)
{
  float_type* cuda_data;
  float_type* cuda_buf_; 
  float_type* cuda_C;
  float_type* cuda_m1; 
  size_t device_freemem=0, device_totalmem=0;
 
  (cudaMemGetInfo(&device_freemem, &device_totalmem));

  size_t cuda_pointN = exp2(floor(log2((double)device_freemem/p->N/sizeof(f_complex)/3)));
  if (cuda_pointN > nN) cuda_pointN=nN;

  int    blocksize = 64;  if (blocksize > cuda_pointN) blocksize=cuda_pointN;
  size_t cuda_piece_size = p->N*cuda_pointN*sizeof(f_complex);


  (cudaMalloc((void**)&cuda_C,        (p->N)*(p->N)*sizeof(float_type ))); 
  (cudaMalloc((void**)&cuda_m1,              (p->N)*sizeof(float_type)));

  (cudaMemcpy(cuda_C,   p->C,      sizeof(float_type)*(p->N)*(p->N), cudaMemcpyHostToDevice));
  (cudaMemcpy(cuda_m1,  p->m1,     sizeof(float_type)*(p->N),        cudaMemcpyHostToDevice));

  if (do_extra_transfer)
  {
  (cudaMalloc((void**)&cuda_data,          cuda_piece_size));
  (cudaMalloc((void**)&cuda_buf_,           cuda_piece_size));

  if (cuda_pointN < nN)
  {
   float_type* data_restrided = (float_type*)malloc_ch(cuda_piece_size); 
   int newstride = cuda_pointN;
 
   for (long i=0; i<nN; i+= cuda_pointN)
   { 
    for (int ni=0; ni<cuda_pointN; ni++)
    for (int nt=0; nt<p->N; nt++) 
    {
	   data_restrided[ni +  2*nt   *newstride]  = real(data[nt*stride + (i+ni)*dist]);
	   data_restrided[ni + (2*nt+1)*newstride]  = imag(data[nt*stride + (i+ni)*dist]);	

	   //data_restrided[ni%(newstride)+(2*nt   + 2*ni/newstride*p->N)*newstride] = real(data[nt*stride + i*dist]);
	   //data_restrided[ni%(newstride)+(2*nt+1 + 2*ni/newstride*p->N)*newstride] = imag(data[nt*stride + i*dist]);
    }
    (cudaMemcpy(cuda_data, data_restrided, cuda_piece_size, cudaMemcpyHostToDevice)) ;
    cuda_qdht_kernel<<< cuda_pointN/blocksize,blocksize >>>(cuda_data, cuda_buf_, p->N,  cuda_C, cuda_m1, 2*newstride, newstride, 1, cuda_pointN);
    (cudaMemcpy(data_restrided, cuda_data, cuda_piece_size, cudaMemcpyDeviceToHost));
   
    for (int ni=0; ni<cuda_pointN; ni++)
    for (int nt=0; nt<p->N; nt++) 
    {
	   //data[nt*stride+ni*dist] = f_complex(data_restrided[ni%newstride+(2*nt +   ni/newstride*p->N)*newstride], \
		                                   data_restrided[ni%newstride+(2*nt+1 + ni/newstride*p->N)*newstride]);
	   data[nt*stride+(i+ni)*dist] = f_complex(data_restrided[ni +  2*nt   *newstride],\
		                                       data_restrided[ni + (2*nt+1)*newstride]); 
    }
   }
   free(data_restrided);
  }
  else
  {
	  (cudaMemcpy(cuda_data, data, cuda_piece_size, cudaMemcpyHostToDevice));
	  cuda_qdht_kernel<<<cuda_pointN/blocksize, blocksize>>>(cuda_data, cuda_buf_, p->N, cuda_C, cuda_m1, 2*stride, 1, 2*dist, cuda_pointN);
	  (cudaMemcpy(data, cuda_data, cuda_piece_size, cudaMemcpyDeviceToHost));
  }

  (cudaFree(cuda_data));
  (cudaFree(cuda_buf_));
  }
  else
  {
        cuda_qdht_kernel<<<cuda_pointN/blocksize, blocksize>>>((float_type*)data, cuda_buf, p->N, cuda_C, cuda_m1, 2*stride, 1, 2*dist, cuda_pointN);
  }

  (cudaFree(cuda_C));
  (cudaFree(cuda_m1)); 
}

__global__ void cuda_qdht_kernel(float_type* data, float_type* buf, size_t N, float_type* C, float_type* m1,  size_t istride, size_t cstride, size_t idist, size_t pointN)
{
	size_t pi      = blockDim.x*blockIdx.x + threadIdx.x; 
	size_t hstride = pointN; 

	data += pi*idist;
  	buf += pi; 

	
	for (size_t i=0; i<N; i++) {data[istride*i]/=m1[i]; data[istride*i+cstride]/=m1[i];}
	for (size_t i=0; i<N; i++) 
	{
  	 buf[2*i*hstride] = 0; buf[(2*i+1)*hstride]=0; 
	 for (size_t j=0; j<N; j++) {float_type cC = C[i+N*j]; buf[2*i*hstride]+= cC*data[istride*j]; buf[(2*i+1)*hstride]+=cC*data[istride*j+cstride];}
	}
	for (size_t i=0; i<N; i++) {data[istride*i]=buf[2*i*hstride]*m1[i]; data[istride*i+cstride]=buf[(2*i+1)*hstride]*m1[i]; }
}



#endif
