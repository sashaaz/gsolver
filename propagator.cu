#include "hip/hip_runtime.h"
#ifdef _ENABLE_CUDA
#include "cuda_extmath.h"
#endif 

#include "ionization.h"


void calculate_NLresponse(f_complex* input, f_complex* output);
void calculate_Hlike_response(f_complex* input, f_complex* output, int N, float_type* Zeff, float_type* alpha);

#ifdef _ENABLE_CUDA
__device__ float_type* cuda_plasma_func;
__constant__ float_type cuda_plasma_factor_re;
__constant__ float_type cuda_plasma_factor_im;

__device__ float_type* cuda_raman_func;
__device__ float_type* cuda_wavenum;
__device__ float_type* cuda_omega;
__device__ float_type* cuda_hodisp;

__constant__ __device__ int cuda_N_T;
__constant__ float_type cuda_TMIN;
__constant__ float_type cuda_TMAX;
__constant__ float_type cuda_TSTEP;

__constant__ float_type cuda_OMEGA_MAX;
__constant__ float_type cuda_OMEGA_MIN;

__constant__ float_type cuda_IONIZATION_POTENTIAL;
__constant__ float_type cuda_RECOMBINATION_TAU;
__constant__ float_type cuda_IONIZATION_POTENIAL;
#ifdef MULTI_LEVEL_IONIZATION
__device__ float_type* cuda_IONIZATION_POTENTIALS;
__device__ int cuda_IONIZATION_LEVEL_N;

#endif 
__constant__ float_type cuda_AMBIENT_CARRIER_DENSITY;
__constant__ float_type cuda_PONDEROMOTIVE_COEFFICIENT;
__constant__ float_type cuda_RAMAN_FRACTION;

__constant__ float_type cuda_N4;
__constant__ float_type cuda_N2;

#ifdef THIRD_HARMONICS
__constant__ float_type cuda_th_factor; 
#endif

__constant__ float_type cuda_AVALANCHE_CROSSSECTION;
__constant__ float_type cuda_GROUP_VELOCITY;
__constant__ float_type cuda_kxstep;
__constant__ float_type cuda_kystep;

__constant__ float_type cuda_wavenum0;
__constant__ float_type cuda_omega0;


__constant__ float_type cuda_NEUTRAL_DENSITY;
#ifdef MULTIPHOTON_IONIZATION
__constant__ int cuda_K_MPI;
__constant__ float_type cuda_BETA_MPI_LN;
#else
__constant__ float_type* cuda_IONIZATION_RATE_LN;
#endif 
__device__  float_type plasma_source_function_device(float_type reA, float_type imA, float_type ro);
__device__  float_type photoionization_function_device(float_type reA, float_type imA, float_type ro);
__device__  float_type photoionization_function_device2(float_type reA, float_type imA, float_type ro1, float_type ro2);
__device__  float_type photoabsorbtion_function_device(float_type reA, float_type imA, float_type ro);
__device__  float_type avalanche_ionization_function_device(float_type reA, float_type imA, float_type ro);
__device__  float_type recombination_function_device(float_type ro);
#ifdef MULTI_LEVEL_IONIZATION
__device__ void photoionization_functionsN_device(float_type reA, float_type imA, float_type* W, int stride);
#endif

__device__  void getpolar(float_type* X, float_type* ro, float_type* phi);

__device__  void calculate_plasmadensity_small_device(float_type* field, float_type* pro, float_type* buf);
__device__  void calculate_plasmadensity_small_device_strided       (float_type* field, float_type* pro, int stride, float_type* buf);
__device__  void calculate_plasmadensity_small_device_strided_2float(float_type* field, float_type* pro, int stride, float_type* buf); 
__device__  void calculate_plasmadensity_losses_small_device(float_type* field, float_type* pro, int stride, int rostride, float_type* loss, float_type* buf);
__device__  float_type calculate_maxplasmadensity_small_device      (float_type* field, float_type* buf); 

//__device__  void calculate_single_NLresponse_kernel        (float_type* input, float_type* output, int N_T, float_type* bufs);
__device__  void calculate_single_NLresponse_kernel_strided(float_type* input, float_type* output, int N_T, float_type* bufs);
//__device__ void calculate_single_NLresponse_kernel_strided(float_type* field, float_type* out, float_type* tempb, int stride);


__global__ void calculate_Lresponse_kernel(float_type* field, float_type* buf1, float_type* kt2_, float_type ZSTEP, size_t N_T, size_t n);
__global__ void calculate_NLresponse_kernel        (f_complex* input, f_complex* output, float_type zstep, int N_T, float_type* bufs);
__global__ void calculate_NLresponse_kernel_strided(f_complex* input, f_complex* output, float_type zstep, int N_T, float_type* bufs, float_type* maA2, float_type* maxNL2);

__global__ void calculate_plasma_2float_kernel (f_complex* input, float_type* output, f_complex* buf);
__global__ void calculate_maxplasma_kernel     (f_complex* input, float_type* output, f_complex* buf); 

#ifdef YUDIN_IVANOV_CORRECTION
__device__ float_type YI_Phi_device(float_type theta, float_type g);
#endif
 
__device__ inline float_type device_hfgaussfilter(float_type omega) {float_type domega = (omega/cuda_OMEGA_MAX-1)/ABSORBTION_LAYER_WIDTH; return (domega<0)?(float_type)1.0:exp_f(-ABSORBTION_LAYER_BETA*domega*domega);} 
__device__ inline float_type device_lfgaussfilter(float_type omega) {float_type domega = (omega*(1-ABSORBTION_LAYER_WIDTH)/cuda_OMEGA_MIN-1); return (domega>0)?(float_type)1.0:exp_f(-ABSORBTION_LAYER_BETA*domega*domega);} 
//__device__ inline float_type device_hfgaussfilter(float_type omega) {float_type domega = (2.0*omega/cuda_OMEGA_MAX-1)/0.99; return exp_p(-pow_p(domega, (float_type)100));} 
//__device__ inline float_type device_lfgaussfilter(float_type omega) {return (float_type)1.0;} 

f_complex* plasma_func_;

bool cuda_do_extra_memtransfer = true; 
bool cuda_use_pinned_memory = false;  

float_type* cuda_field; 
float_type* cuda_buf1; 
float_type* cuda_buf2;

float_type* cuda_bufs;
float_type* cuda_maxA2;
float_type* cuda_maxNL2;

 size_t cuda_device_freemem=0, cuda_device_totalmem=0;
 size_t cuda_pointN = 0, cuda_piece_size = 0; 
 size_t cuda_blocksize = 64, cuda_maxbuf_len = 0;
#endif 

void calculate_NLstep(float_type* maxI, float_type* maxNL2);
void calculate_Lstep ();


void propagator()
{

#ifndef _SILENCE
#ifdef _SHOW_EVERY_STEP
	if (ISMASTER) printf("\nCalculating nonlinear responses...");fflush(stdout);
#endif
#endif
	while (true)
	{
	 float_type max_NL2 = 0;
	 float_type max_A2   = 0;


         calculate_NLstep(&max_A2, &max_NL2);

	 float_type max_NL2_0 = max_NL2, max_A2_0 = max_A2;
	 MPI_Allreduce(&max_NL2_0, &max_NL2, 1, MPI_FLOAT_TYPE, MPI_MAX, MPI_COMM_WORLD);  
	 MPI_Allreduce(&max_A2_0,  &max_A2,  1, MPI_FLOAT_TYPE, MPI_MAX, MPI_COMM_WORLD);  

         bool repeatstep = false; 	

#ifndef _ODE_EULER
	 if (ZSTEP*ZSTEP*max_NL2 >      MAX_TOLERANCE*MAX_TOLERANCE*max_A2) {repeatstep = true; STEP_N++;}
#endif
	 ZSTEP = 0.5*MAX_TOLERANCE*sqrt(max_A2/max_NL2);
#ifdef _UNIAXIAL_FINITE_DIFFERENCE
	ZSTEP = min(ZSTEP, LIN_ZSTEP);
#endif

#ifdef _SHOW_EVERY_STEP
	 if (ISMASTER) { printf("Done. (%5.2fs)Z=%e, max_A2=%e, max_NL2=%e, Zstep=%e", MPI_Wtime()-TIME_START,CURRENT_Z, max_A2, max_NL2, ZSTEP); fflush(stdout);}
#endif
		
	 if (ZSTEP < MINSTEP_RATIO*(ZNET[n_Z]-ZNET[n_Z-1]) || !(ZSTEP > 0)) 
	 {
	  printf("[%d]: max_A2=%e, max_NL2=%e, Zstep=%e", PROCESS_RANK, max_A2, max_NL2, ZSTEP);
	  throw "Minimum Z step size reached, collapse is possible!";
	 }	 
	ZSTEP = min(ZSTEP, ZNET[n_Z]-CURRENT_Z);
	
	 if (!repeatstep)
	 {

#ifndef _UNIAXIAL_FINITE_DIFFERENCE
#ifndef NO_DIFFRACTION
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("\nRunning Fourier transform for nonlinear response array... "); fflush(stdout); }
 #endif 
#ifdef _UNIAXIAL
	#ifdef _ENABLE_CUDA
          if (!cuda_do_extra_memtransfer) ht_run((f_complex*)cuda_buf1, (f_complex*)cuda_bufs); 
	  else
        #endif
	  ht_run(BIGBUFFER1, FIELD);
		
       	 
#else
       fftwt_execute(FFT_FWPLAN_XY); 
#endif
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("Done"); fflush(stdout); }
 #endif 

#endif
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("\nCalculating field change during propagation step... "); fflush(stdout); }
 #endif 
 
       calculate_Lstep(); 
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("Done."); fflush(stdout); }
 #endif 
 

#ifndef NO_DIFFRACTION
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("\nRunning Fourier transform for field array... "); fflush(stdout); }
 #endif 
#ifdef _UNIAXIAL
	 #ifdef _ENABLE_CUDA
          if (!cuda_do_extra_memtransfer) 
          {
           ht_run((f_complex*)cuda_field, (f_complex*)cuda_bufs);
           (hipMemcpy(FIELD, cuda_field, cuda_piece_size, hipMemcpyDeviceToHost));
          }
	  else
        #endif
	ht_run(FIELD, BIGBUFFER1);	
#else
       fftwt_execute(FFT_BWPLAN_XY);
       for (size_t i=0; i<N_T*MY_NX*MY_NY; i++) FIELD[i] /= N_X*N_Y;       			
#endif
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("\nDone. "); fflush(stdout); }
 #endif 
#endif 
#else
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("\nCalculating field change during propagation step using finite differences... "); fflush(stdout); }
 #endif 
	
	calculate_Lstep_UAFD();
 #ifdef _SHOW_EVERY_STEP
	if (ISMASTER) {printf("Done."); fflush(stdout); }
 #endif 
#endif
	if (APERTURE_N > 0) if (CURRENT_Z >= APERTURE_Z[APERTURE_N - 1])
	{
	 for (int nx=0; nx<MY_NX; nx++)
	 for (int ny=0; ny<MY_NY; ny++)
	 {
 #ifdef _UNIAXIAL
 #ifndef _UNIAXIAL_FINITE_DIFFERENCE
	   float_type R = HT_PLAN->x_n(nx+MY_NXstart)*XMAX;
 #else
	   float_type R = XMAX*(nx+MY_NXstart)/N_X;
 #endif
 #else
	   float_type x = XMIN+nx*XSTEP, y = YMIN+(ny+MY_NYstart)*YSTEP;
	   float_type R = sqrt(x*x+y*y);
 #endif	
	   if (R > APERTURE_R[APERTURE_N - 1]) for (int nw=0; nw<N_T; nw++) FIELD[nw + N_T*(nx+MY_NX*ny)]=0.0; 
	   
 	 }

#ifndef _UNIAXIAL_FINITE_DIFFERENCE		
	 memcpy(BIGBUFFER2, FIELD, MY_NX*MY_NY*N_T*sizeof(f_complex)); 
#ifdef _UNIAXIAL
	 ht_run(BIGBUFFER2, BIGBUFFER1);
#else 
	 fftwt_mpi_execute_dft(FFT_FWPLAN_XY, (fftwt_complex*)BIGBUFFER2, (fftwt_complex*)BIGBUFFER2);
#endif
#endif
	 if (ISMASTER) {printf("\n Aperture #%d of radius %g is applied", APERTURE_N, APERTURE_R[APERTURE_N-1]); fflush(stdout);}
	 APERTURE_N--; 
	}

	   return; 
	 }
	}
}

void calculate_NLstep(float_type* maxI, float_type* maxNL2)
{
#ifdef _ENABLE_CUDA
 f_complex* cuda_field_piece;
 float_type* maxA2buf, *maxNL2buf; 
 #ifdef _ODE_RK4
    int bufs_size_factor = 10;
  #endif
  #ifdef _ODE_HEUN
    int bufs_size_factor = 8;
  #endif
  #ifdef _ODE_EULER
    int bufs_size_factor = 7;
  #endif
  
   
  if (STEP_N == 0) 
  {

    int devicenum = 0; 
    (hipGetDevice(&devicenum));
    (hipMemGetInfo(&cuda_device_freemem, &cuda_device_totalmem));


     cuda_pointN = (int)pow(2.0,floor_(log2((double)(cuda_device_freemem/N_T/(bufs_size_factor+1)/sizeof(f_complex)))));
     if (cuda_pointN > MY_NX*MY_NY) cuda_pointN=MY_NX*MY_NY;
  
     if (cuda_blocksize > cuda_pointN) cuda_blocksize=cuda_pointN;
     cuda_piece_size = N_T*cuda_pointN*sizeof(f_complex);
     cuda_maxbuf_len = sizeof(float_type)*cuda_pointN; 


     printf("\n[%d]: Launching on CUDA device %d. It has %ld bytes of memory free. Allocating for %ld points", PROCESS_RANK, devicenum, cuda_device_freemem, cuda_pointN); fflush(stdout);
  
    if (cuda_device_freemem > N_T*MY_NX*MY_NY*(bufs_size_factor+3)*sizeof(f_complex) &&false)
    {
     cuda_do_extra_memtransfer = false;

    if (ISMASTER) printf("\n Storing all data in CUDA device memory."); 
     
    (hipMalloc((void**)&cuda_field, cuda_piece_size)); 
    (hipMalloc((void**)&cuda_buf1, cuda_piece_size)); 
    (hipMalloc((void**)&cuda_buf2, cuda_piece_size));
  
    (hipMemcpy(cuda_field, FIELD,       cuda_piece_size, hipMemcpyHostToDevice)); 
    (hipMemcpy(cuda_buf2,  BIGBUFFER2,  cuda_piece_size, hipMemcpyHostToDevice)); 

	
    
    (hipMalloc((void**)&cuda_bufs,        bufs_size_factor*cuda_piece_size));
    (hipMalloc((void**)&cuda_maxA2,       cuda_maxbuf_len)); 
    (hipMalloc((void**)&cuda_maxNL2,      cuda_maxbuf_len)); 
   }
   if (cuda_use_pinned_memory) 
   {	
      if (ISMASTER) printf("\n Using page-locked memory. ");
      (hipHostRegister(FIELD, cuda_piece_size, 0)); 
      (hipHostRegister(BIGBUFFER1, cuda_piece_size, 0)); 
      (hipHostRegister(BIGBUFFER2, cuda_piece_size, 0)); 
   }   
  }

 maxA2buf = (float_type*)malloc_ch(sizeof(float_type)*cuda_pointN); maxNL2buf = (float_type*)malloc_ch(sizeof(float_type)*cuda_pointN); 

 if (cuda_do_extra_memtransfer)
 {
   (hipMalloc((void**)&cuda_field_piece,                  cuda_piece_size));
   (hipMalloc((void**)&cuda_bufs,        bufs_size_factor*cuda_piece_size));
 
    (hipMalloc((void**)&cuda_maxA2,       cuda_maxbuf_len)); 
    (hipMalloc((void**)&cuda_maxNL2,      cuda_maxbuf_len)); 
 
 for (long i=0; i<MY_NX*MY_NY; i+= cuda_pointN)
 {
  (hipMemcpy(cuda_field_piece, FIELD+i*N_T,      cuda_piece_size, hipMemcpyHostToDevice)) ;
  calculate_NLresponse_kernel_strided<<< cuda_pointN/cuda_blocksize,cuda_blocksize >>>(cuda_field_piece, cuda_field_piece, ZSTEP, N_T, cuda_bufs, cuda_maxA2, cuda_maxNL2);
  (hipMemcpy(BIGBUFFER1+i*N_T, cuda_field_piece, cuda_piece_size, hipMemcpyDeviceToHost)) ;
  (hipMemcpy(maxA2buf,  cuda_maxA2,  cuda_maxbuf_len, hipMemcpyDeviceToHost)); 
  (hipMemcpy(maxNL2buf, cuda_maxNL2, cuda_maxbuf_len, hipMemcpyDeviceToHost)); 
  //for (int np=0; np<cuda_pointN; np++) { (*maxI) = max((*maxI), maxA2buf[np]); (*maxNL2) = max((*maxNL2), maxNL2buf[np]);}
 }
  for (long i=0; i<MY_NX*MY_NY*N_T; i++)
  {
   (*maxI)   = max((*maxI),   abs2(FIELD[i])); 
   (*maxNL2) = max((*maxNL2), abs2(BIGBUFFER1[i])); 
  }
 }
 
 else
 {
  calculate_NLresponse_kernel_strided<<< cuda_pointN/cuda_blocksize,cuda_blocksize >>>((f_complex*)cuda_field, (f_complex*)cuda_buf1, ZSTEP, N_T, cuda_bufs, cuda_maxA2, cuda_maxNL2);
  (hipMemcpy(maxA2buf,  cuda_maxA2,  cuda_maxbuf_len, hipMemcpyDeviceToHost)); 
  (hipMemcpy(maxNL2buf, cuda_maxNL2, cuda_maxbuf_len, hipMemcpyDeviceToHost)); 
  for (int np=0; np<cuda_pointN; np++) { (*maxI) = max((*maxI), maxA2buf[np]); (*maxNL2) = max((*maxNL2), maxNL2buf[np]);}
 }
 //for (int i=0; i<MY_NX*MY_NY*N_T; i++) {(*maxI) = max((*maxI), abs2(FIELD[i])); (*maxNL2)=max((*maxNL2), abs2(BIGBUFFER1[i])); }



 if (cuda_do_extra_memtransfer)
 {
  (hipFree(cuda_maxA2)); 
  (hipFree(cuda_maxNL2)); 
  (hipFree(cuda_field_piece)); 
  (hipFree(cuda_bufs)); 
 }
 free(maxA2buf); free(maxNL2buf);
#else

  float_type max_I = 0, max_NL2 = 0;
  for (int ny=0; ny<MY_NY; ny++) for (int nx=0; nx<MY_NX; nx++)
  {
	   int ofs0 = N_T*(nx+MY_NX*ny);
#ifdef NONLINEARITY_ON
 #ifdef _ODE_EULER
	  calculate_NLresponse(FIELD+ofs0, BIGBUFFER1+ofs0);
 #endif

   #ifdef _ODE_RK4
	  calculate_NLresponse(FIELD+ofs0, NL_OUTPUT1);

	  for (int nt=0; nt<N_T; nt++) NL_INPUT[nt] = FIELD[ofs0+nt] + ZSTEP*(float_type)0.5*NL_OUTPUT1[nt];
	  calculate_NLresponse(NL_INPUT, NL_OUTPUT2);

	  for (int nt=0; nt<N_T; nt++) NL_INPUT[nt] = FIELD[ofs0+nt] + ZSTEP*(float_type)0.5*NL_OUTPUT2[nt];
	  calculate_NLresponse(NL_INPUT, NL_OUTPUT3);

	  for (int nt=0; nt<N_T; nt++) NL_INPUT[nt]= FIELD[ofs0+nt]  + ZSTEP*NL_OUTPUT3[nt];
	  calculate_NLresponse(NL_INPUT, NL_OUTPUT4); 
 #endif 

 #ifdef _ODE_HEUN
	  calculate_NLresponse(FIELD+ofs0, NL_OUTPUT1);

	  for (int nt=0; nt<N_T; nt++) NL_INPUT[nt] = FIELD[ofs0+nt] + ZSTEP*NL_OUTPUT1[nt];
	  calculate_NLresponse(NL_INPUT, NL_OUTPUT2);
 #endif

#endif

	  for (int nt=0; nt<N_T; nt++)
	  {
#ifdef NONLINEARITY_ON
 #ifdef _ODE_RK4
	   BIGBUFFER1[ofs0+nt] = (NL_OUTPUT1[nt] + (float_type)2.0*NL_OUTPUT2[nt] + (float_type)2.0*NL_OUTPUT3[nt] + NL_OUTPUT4[nt])/(float_type)6.0;
 #endif
 #ifdef _ODE_HEUN
	   BIGBUFFER1[ofs0+nt] = (NL_OUTPUT1[nt] + NL_OUTPUT2[nt])/(float_type)2.0;
 #endif
#else
       BIGBUFFER1[ofs0+nt] = 0.0;
#endif
 	   max_NL2 =  max(max_NL2,  abs2(BIGBUFFER1[ofs0+nt]));
	   max_I   =  max(max_I,    abs2(FIELD     [ofs0+nt]));
	  } 
  }
  (*maxI)=max_I; (*maxNL2)=max_NL2;
#endif 
}


#ifdef _UNIAXIAL_FINITE_DIFFERENCE

void calculate_Lstep_UAFD()
{
  f_complex j = f_complex(0,1);
  MPI_Status mpistatus; 

  for (int nw=0; nw<N_T;   nw++) 
  {
    f_complex* kappa = BIGBUFFER2      +nw*(MY_NX); 
    f_complex* khi   = BIGBUFFER2+(N_T+nw)*(MY_NX);  
    f_complex khibuf, kappabuf, fieldbuf;
    //calculate alpha, beta and gamma and, finally, khi and kappa.
    if (PROCESS_RANK > 0) 
    {
//	printf("\n[%d]:Revceiving kappa and khi with tag %d from %d...", PROCESS_RANK, nw, PROCESS_RANK-1); fflush(stdout);
	MPI_Recv(&khibuf,   2, MPI_FLOAT_TYPE, PROCESS_RANK-1, nw, MPI_COMM_WORLD, &mpistatus);
	MPI_Recv(&kappabuf, 2, MPI_FLOAT_TYPE, PROCESS_RANK-1, nw, MPI_COMM_WORLD, &mpistatus);
	MPI_Recv(&fieldbuf, 2, MPI_FLOAT_TYPE, PROCESS_RANK-1, nw, MPI_COMM_WORLD, &mpistatus);
//	printf("[%d]:Success! ", PROCESS_RANK); fflush(stdout);

	float_type rm = XMAX*(MY_NXstart-1)/N_X, r = XMAX*(MY_NXstart)/N_X, rp = XMAX*(1+MY_NXstart)/N_X;
	float_type hm = r-rm, hp=rp-r; 
	
	f_complex S = -j*ZSTEP/(real(WAVENUMBER[nw]))/(float_type)4.0;
	f_complex A = S/hm*((float_type)2.0/(rp-rm)-(float_type)1.0/(r+rm));	
        f_complex C = S/hp*((float_type)2.0/(rp-rm)+(float_type)1.0/(rp+r));
	f_complex B = S*((float_type)1.0/hm/(r+rm) - (float_type)1.0/hp/(rp+r) - (float_type)2.0/hm/hp) - (float_type)1.0; 
 
	f_complex D = -A*fieldbuf - (B+(float_type)2.0)*FIELD[nw] - C*FIELD[nw+N_T] - BIGBUFFER1[nw]*ZSTEP;
   	
	khi[0]    = (D-A*khibuf)/(A*kappabuf+B); 
        kappa[0]  = -C          /(A*kappabuf+B); 
    }    
    else {khi[0]=0; kappa[0]=1;}

    for (int nx=1; nx<MY_NX; nx++)
    {
	float_type rm = XMAX*(nx+MY_NXstart-1)/N_X, r = XMAX*(nx+MY_NXstart)/N_X, rp = XMAX*(nx+1+MY_NXstart)/N_X;
	float_type hm = r-rm, hp=rp-r; 
	size_t ofs = nw+N_T*nx;
#ifndef NO_SPACE_TIME_FOCUSING
	float_type k = real(WAVENUMBER[nw]);
#else 
	float_type k = real(WAVENUMBER0); 
#endif
	f_complex S = -j*ZSTEP/(real(WAVENUMBER[nw]))/(float_type)4.0;
	f_complex A = S/hm*((float_type)2.0/(rp-rm)-(float_type)1.0/(r+rm));	
        f_complex C = S/hp*((float_type)2.0/(rp-rm)+(float_type)1.0/(rp+r));
	f_complex B = S*((float_type)1.0/hm/(r+rm) - (float_type)1.0/hp/(rp+r) - (float_type)2.0/hm/hp) - (float_type)1.0; 
 
	f_complex D = -A*FIELD[ofs-N_T] - (B+(float_type)2.0)*FIELD[ofs] - C*FIELD[ofs+N_T] - BIGBUFFER1[ofs]*ZSTEP;
   	
	khi[nx]    = (D-A*khi[nx-1])/(A*kappa[nx-1]+B); 
        kappa[nx]  =-C              /(A*kappa[nx-1]+B);
    }
    if (PROCESS_RANK < PROCESS_N-1) 
    {
//	printf("\n[%d]:Sending kappa and khi with tag %d to %d...", PROCESS_RANK, nw, PROCESS_RANK+1); fflush(stdout);
    	MPI_Send(khi+MY_NX-1,            2, MPI_FLOAT_TYPE, PROCESS_RANK+1, nw, MPI_COMM_WORLD);
    	MPI_Send(kappa+MY_NX-1,          2, MPI_FLOAT_TYPE, PROCESS_RANK+1, nw, MPI_COMM_WORLD);
	MPI_Send(FIELD+nw+N_T*(MY_NX-1), 2, MPI_FLOAT_TYPE, PROCESS_RANK+1, nw, MPI_COMM_WORLD); 
//	printf("[%d]:Success!", PROCESS_RANK); fflush(stdout);
    }	
  }

  for (int nw=0; nw<N_T; nw++)
  {
    f_complex* kappa = BIGBUFFER2      +nw*(MY_NX);  
    f_complex* khi   = BIGBUFFER2+(N_T+nw)*(MY_NX);  
    if (PROCESS_RANK < PROCESS_N-1) MPI_Recv(FIELD+nw+MY_NX*N_T, 2, MPI_FLOAT_TYPE, PROCESS_RANK+1, nw, MPI_COMM_WORLD, &mpistatus);
    else FIELD[nw+MY_NX*N_T] = 0;//-(khi[MY_NX-1])/(kappa[MY_NX-1]-(float_type)1.0);  //Dierichlet boundary outer boundary condition

    for (int nx=MY_NX-1; nx>=0; nx--)
    {
      size_t ofs = nw+N_T*nx; 
      FIELD[ofs] = FIELD[ofs+N_T]*kappa[nx]+khi[nx];	
    }
    if (PROCESS_RANK > 0) MPI_Send(FIELD+nw, 2, MPI_FLOAT_TYPE, PROCESS_RANK-1, nw, MPI_COMM_WORLD);
  }
  
  for (int nw=0; nw<N_T; nw++) for (int nx=0; nx<=MY_NX; nx++) FIELD[nw+N_T*nx] *= exp(ZSTEP*(-j*HO_DISPERSION[nw]+imag(WAVENUMBER[nw])));
}	

#endif


void calculate_Lstep()
{
#ifdef _ENABLE_CUDA

 float_type* kt2 = (float_type*)malloc_ch(sizeof(float_type)*N_Y*MY_NX_FT);
 float_type* cuda_kt2;
 if (cuda_do_extra_memtransfer) (hipMalloc((void**)&cuda_kt2, sizeof(float_type)*N_Y*MY_NX_FT));
 else cuda_kt2 = cuda_bufs;

#ifndef _UNIAXIAL 
 float_type kxstep = (float_type)2.0*M_PI/(XMAX-XMIN); 
 float_type kystep = (float_type)2.0*M_PI/(YMAX-YMIN); 
#endif 

 for (size_t nx=0; nx<MY_NX_FT; nx++) for (size_t ny=0; ny<N_Y; ny++)
 {
#ifdef _UNIAXIAL
   float_type kt = 2*M_PI*HT_PLAN->x_n(nx+MY_NXstart_FT)*HT_PLAN->getNf()/XMAX;
   kt2[nx+MY_NX_FT*ny] = kt*kt;
#else
  size_t nx_g = nx+MY_NXstart_FT;   
  float_type kx=0; if (nx_g<=N_X/2) kx=kxstep*nx_g; else kx=-kxstep*(N_X-nx_g);
  float_type ky=0; if (ny<=N_Y/2)   ky=kystep*ny;   else ky=-kystep*(N_Y-ny);
  kt2[ny+N_Y*nx] = kx*kx+ky*ky;
#endif 
 }
 (hipMemcpy(cuda_kt2, kt2, sizeof(float_type)*N_Y*MY_NX_FT, hipMemcpyHostToDevice));
 free(kt2);

 float_type* cuda_field_piece, *cuda_buf_piece; 

 if (cuda_do_extra_memtransfer)
 {
  (hipMalloc((void**)&cuda_field_piece,    cuda_piece_size));
  (hipMalloc((void**)&cuda_buf_piece,      cuda_piece_size));
  for (size_t i=0; i<N_Y*MY_NX_FT; i+= cuda_pointN)
  {
   (hipMemcpy(cuda_field_piece, BIGBUFFER2+i*N_T,      cuda_piece_size, hipMemcpyHostToDevice)) ;
   (hipMemcpy(cuda_buf_piece,   BIGBUFFER1+i*N_T, cuda_piece_size, hipMemcpyHostToDevice)) ;
   calculate_Lresponse_kernel<<< cuda_pointN/cuda_blocksize,cuda_blocksize >>>(cuda_field_piece, cuda_buf_piece, cuda_kt2, ZSTEP, N_T, i);
   (hipMemcpy(FIELD+i*N_T,      cuda_field_piece, cuda_piece_size, hipMemcpyDeviceToHost));
   (hipMemcpy(BIGBUFFER2+i*N_T, cuda_field_piece, cuda_piece_size, hipMemcpyDeviceToHost));
  }
 
 (hipFree(cuda_field_piece)); 
 (hipFree(cuda_buf_piece)); 
 (hipFree(cuda_kt2));
 }
 else 
 {
   calculate_Lresponse_kernel<<< cuda_pointN/cuda_blocksize,cuda_blocksize >>>(cuda_buf2, cuda_buf1, cuda_kt2, ZSTEP, N_T, 0);
   (hipMemcpy(cuda_field, cuda_buf2, cuda_piece_size, hipMemcpyDeviceToDevice));   
 //  (hipMemcpyAsync(BUGBUFFER2, cuda_buf2, cuda_piece_size, hipMemcpyDeviceToHost)); 
 }
#else

#ifndef _UNIAXIAL 
 float_type kxstep = (float_type)2.0*M_PI/(XMAX-XMIN); 
 float_type kystep = (float_type)2.0*M_PI/(YMAX-YMIN); 
#endif 

 f_complex j=f_complex(0,1);
 for (size_t nx=0; nx<MY_NX_FT; nx++) for (size_t ny=0; ny<N_Y; ny++)
 {
   float_type kt2 = 0;
#ifdef _UNIAXIAL
   float_type kt = 2*M_PI*HT_PLAN->x_n(nx+MY_NXstart_FT)*HT_PLAN->getNf()/XMAX;
   kt2 = kt*kt; 
#else
  size_t nx_g = nx+MY_NXstart_FT;   
  float_type kx=0; if (nx_g<=N_X/2) kx=kxstep*nx_g; else kx=-kxstep*(N_X-nx_g);
  float_type ky=0; if (ny<=N_Y/2)   ky=kystep*ny;   else ky=-kystep*(N_Y-ny);
  kt2 = kx*kx+ky*ky;
#endif 

  for (size_t nw = 0; nw<N_T;     nw++)
  {
    size_t ofs = (nw + N_T*(ny+N_Y*nx));
	float_type w = OMEGA[nw];
#ifndef NO_SPACE_TIME_FOCUSING
	f_complex  k = WAVENUMBER[nw]; 
#else 
	f_complex  k = WAVENUMBER0;
#endif

	if (w<OMEGA_MIN || w > OMEGA_MAX)     {FIELD[ofs]=0.0;continue;}
#ifndef NO_DIFFRACTION
	if (kt2>MAX_KT2*real(k*k)) {FIELD[ofs]=0.0;continue;} 
#endif 
    //float_type f = lfgaussfilter(w)*hfgaussfilter(w);

#ifndef NO_DIFFRACTION
	float_type k1 = real(k), k2=imag(k);
#ifdef PARABOLICAL_DIFRACTION
	BIGBUFFER2[ofs] += ZSTEP*BIGBUFFER1[ofs];
	FIELD[ofs] = (BIGBUFFER2[ofs]*exp(ZSTEP*(j*(kt2/2/k1-HO_DISPERSION[nw])-k2)));// + ZSTEP*BIGBUFFER1[ofs]);
#else
    //f_complex kz = sqrt(k*k - kt2);   	     
	float_type sqHO = sqrtHO(-kt2/k1/k1);
	BIGBUFFER2[ofs] += ZSTEP*BIGBUFFER1[ofs]/(1+sqHO);
	FIELD[ofs] = (BIGBUFFER2[ofs]*exp(ZSTEP*(j*(-HO_DISPERSION[nw]-sqHO*k1)+k2)));// + ZSTEP*BIGBUFFER1[ofs]/(1+sqHO));
#endif
#else
	FIELD[ofs] = BIGBUFFER2[ofs]*exp(-ZSTEP*j*HO_DISPERSION[nw]) + ZSTEP*BIGBUFFER1[ofs];
#endif
	BIGBUFFER2[ofs]=FIELD[ofs];
  }
 }

#endif
}


void calculate_NLresponse(f_complex* input, f_complex* output)  
{
	//This function calculates nonlinear response at one spatial net point.
	f_complex j = f_complex(0,1); 

#ifndef NONLINEARITY_ON
        for (int nt=0; nt<N_T; nt++) output[nt] = 0;
        return;
#endif

#ifndef H_LIKE_RESPONSE
	fftwt_execute_dft(FFT_BWPLAN_T, (fftwt_complex*)input, (fftwt_complex*)FIELD_REAL_SMALL);
	fftwt_Nnormalize(1, FIELD_REAL_SMALL);
#ifndef NO_PLASMARESPONSE
	calculate_plasmadensity_losses_small(FIELD_REAL_SMALL, (float_type*)NL_SMALLBUFFER3, 2, NL_SMALLBUFFER1, NL_SMALLBUFFER4);           //calculate plasma density at this point
	

	//float_type tstep = (TMAX - TMIN)/N_T;

	for (int nt=0; nt<N_T; nt++)
	{
	 float_type ro = real(NL_SMALLBUFFER3[nt]);
#ifndef PLASMA_FULL_DISPERSION
	 f_complex E = FIELD_REAL_SMALL[nt];
#ifdef PLASMA_DISPERSION
	 NL_SMALLBUFFER3[nt] = ro*E;													 //multiply plasma density by field
 #else
	 f_complex Fro = ro*PLASMA_FACTOR, sFro = 0;
	 if (fabs(real(Fro))<0.1) sFro = sqrtHO(-Fro);
	 else sFro = sqrt((float_type)1.0-Fro)-(float_type)1.0;

	 NL_SMALLBUFFER1[nt] +=  - j*WAVENUMBER0*sFro*E;
 #endif 
#else
	 f_complex w0  = exp((float_type)2.0*(float_type)M_PI*j*((float_type)nt)/((float_type)N_T));
	 f_complex M   = 1.0/N_T;
	 for (int nw=0; nw<N_T; nw++)
	 { 
		f_complex Fro_ = PLASMA_FUNC[nw]*ro, sFro_ = 0;   
		if (fabs(real(Fro_))<0.1) sFro_ = sqrtHO(+Fro_);
		else sFro_ = sqrt((float_type)1.0+Fro_)-(float_type)1.0;

		NL_SMALLBUFFER1[nt]+=-j*WAVENUMBER[nw]*sFro_*input[nw]*M;
		M*=w0;
	 } 
	
#endif 
	}
	
 #ifdef PLASMA_DISPERSION
	fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER1, (fftwt_complex*)NL_SMALLBUFFER2);    //These two lines execute forward Fourier transform 1->2 and 3->1 
	fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER3, (fftwt_complex*)NL_SMALLBUFFER1);    //i.e. for ro*E and for gamma*E, where ro is plasma density, E - field and gamma - PA losses

	for (int nw=0; nw<N_T;   nw++)
	{		
	 NL_SMALLBUFFER1[nw] *= PLASMA_FUNC[nw];
	 NL_SMALLBUFFER2[nw] += NL_SMALLBUFFER1[nw];            //now NL_SMALLBUFFER2 contains all ionization responses.
	}
 #else
	fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER1, (fftwt_complex*)NL_SMALLBUFFER2);
 #endif 
#else 
	for (int nw=0; nw<N_T; nw++) {NL_SMALLBUFFER2[nw]=0; NL_SMALLBUFFER4[nw]=1.0;}
#endif
	
    if (NONLIN_REFRINDEX != 0)
    {
	 for (int nt=0; nt<N_T;  nt++) NL_SMALLBUFFER3[nt]  = abs2(FIELD_REAL_SMALL[nt]);  //put intensity into NL_SMALLBUFFER2 
	 if (RAMAN_FRACTION > 0.001)					     
	 {
	  //Calculate delayed nonlinearity response using intensity Fourier-transform
	  fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER3, (fftwt_complex*)NL_SMALLBUFFER1);
	  for (int nw=0; nw<N_T; nw++) NL_SMALLBUFFER1[nw] *= RAMAN_FUNCTION[nw];
	  fftwt_execute_dft(FFT_BWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER1, (fftwt_complex*)NL_SMALLBUFFER3);
	  fftwt_Nnormalize(1,NL_SMALLBUFFER3);
	  for (int nt=0; nt<N_T; nt++) { f_complex  E=FIELD_REAL_SMALL[nt]; NL_SMALLBUFFER3[nt]*=E;} 
	}
	else
	for (int nt=0; nt<N_T; nt++) { f_complex  E=FIELD_REAL_SMALL[nt]; NL_SMALLBUFFER3[nt]*=E;}

	fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER3, (fftwt_complex*)NL_SMALLBUFFER1); 
	for (int nw=0; nw<N_T; nw++) { NL_SMALLBUFFER2[nw] += -j*KERR_PROFILE[nw]*NL_SMALLBUFFER1[nw];}
	
#ifdef THIRD_HARMONICS
	for (int nt=0; nt<N_T; nt++) 
 	{
      	 float_type carrier_phase = (OMEGA0 == OMEGA[0])?(-2.0*OMEGA0*(TMIN+nt*(TMIN-TMAX)/N_T)):0;
 	 f_complex  E=FIELD_REAL_SMALL[nt]; NL_SMALLBUFFER3[nt]=E*E*E*exp(j*carrier_phase);
	}
	fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER3, (fftwt_complex*)NL_SMALLBUFFER1); 
	for (int nw=0; nw<N_T; nw++) { NL_SMALLBUFFER2[nw] += -j*KERR_TH_PROFILE[nw]*NL_SMALLBUFFER1[nw];}
#endif
   }
   
   if (NONLIN_REFRINDEX4 != 0)
   {
	for (int nt=0; nt<N_T; nt++) 
	{ f_complex  E=FIELD_REAL_SMALL[nt]; float_type I = abs2(E);
          NL_SMALLBUFFER3[nt]=I*I*E;
#ifdef THIRD_HARMONICS 
      	 float_type carrier_phase = (OMEGA0 == OMEGA[0])?(-2.0*OMEGA0*(TMIN+nt*(TMIN-TMAX)/N_T)):0;
          NL_SMALLBUFFER3[nt] += TH_FACTOR*(I*E*E*E*exp(j*carrier_phase)/2.0 + E*E*E*E*E*exp(2.0*j*carrier_phase)); 
#endif
        }
	fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER3, (fftwt_complex*)NL_SMALLBUFFER1); 
#ifndef NO_SHOCK
	for (int nw=0; nw<N_T; nw++) { NL_SMALLBUFFER2[nw] += -j*OMEGA[nw]/LIGHT_VELOCITY*NL_SMALLBUFFER3[nw];}		
#else
	for (int nw=0; nw<N_T; nw++) { NL_SMALLBUFFER2[nw] += -j*OMEGA[0]/LIGHT_VELOCITY*NL_SMALLBUFFER3[nw];}		
	
#endif
   }

/*
	 for (int nt=0; nt<N_T;  nt++) 
	 {
	  f_complex E = FIELD_REAL_SMALL[nt];
	  float_type I = abs2(E);
#ifndef THIRD_HARMONICS
	  NL_SMALLBUFFER3[nt] *= E*NL_SMALLBUFFER4[nt];
	  NL_SMALLBUFFER3[nt] += NONLIN_REFRINDEX4*I*I*E*NL_SMALLBUFFER3[nt];

#else
      	  float_type carrier_phase = (OMEGA0 == OMEGA[0])?(-2.0*OMEGA0*(TMIN+nt*(TMIN-TMAX)/N_T)):0;
	  if (NL_SMALLBUFFER3[nt]) = NL

	  NL_SMALLBUFFER3[nt] = NONLIN_REFRINDEX*(NL_SMALLBUFFER3[nt] + (1-RAMAN_FRACTION)*((float_type)(1.0/3.0))*TH_FACTOR*E*E*exp(j*carrier_phase))*E*NL_SMALLBUFFER4[nt];
	  NL_SMALLBUFFER3[nt] += NONLIN_REFRINDEX4*(I*I + TH_FACTOR*((float_type)(1.0/2.0)*exp(j*carrier_phase)*I*E*E + (float_type)(1.0/10.0)*exp((float_type)2.0*j*carrier_phase)*E*E*E*E))*E;
#endif
	 }

	 fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER3, (fftwt_complex*)NL_SMALLBUFFER1);

	 for (int nw=0; nw<N_T; nw++) 
	 {
#ifndef NO_SHOCK
	    float_type w = OMEGA[nw];
#else
	    float_type w = OMEGA0;
#endif
	    output[nw]	= -j*(KERR_PROFILE*NL_SMALLBUFFER1[nw]) + NL_SMALLBUFFER2[nw];
         }

    }
    else*/
   for (int nw=0; nw<N_T; nw++) output[nw] = NL_SMALLBUFFER2[nw];
#else
   float_type Zeff[2] = {1, 1}; 
   float_type alpha[2] = {1, 1}; 

   calculate_Hlike_response(input, output, 2, Zeff, alpha); 
#endif
 //  for (int nw=0; nw<N_T; nw++) {float_type w_ = OMEGA[nw]; output[nw]*=hfgaussfilter(w_)*lfgaussfilter(w_);}

}


void calculate_Hlike_response(f_complex* input, f_complex* output, int N, float_type* Zeff, float_type* alpha)
{
 float_type Xmax = 20; 
 int Nx   = 256; 
 float_type dt0 = 0.2; 
 f_complex j = f_complex(0,1); 

 int Nt   = (TMAX-TMIN)/ATOMIC_TIME/dt0; 
 int tfac = (int)exp2(ceil(log2((float_type)Nt/(float_type)N_T))); 
 Nt = N_T*tfac; 

 float_type dt =(TMAX-TMIN)/Nt/ATOMIC_TIME; 
 
 f_complex* E  = (f_complex*)malloc_ch(Nt*sizeof(f_complex)); 
 f_complex* Er = (f_complex*)malloc_ch(Nt*sizeof(f_complex)); 
 
 memcpy(E,            input,        N_T/2*sizeof(f_complex)); 
 memcpy(E+(Nt-N_T/2), input+N_T/2,  N_T/2*sizeof(f_complex)); 
 for (int nt=N_T/2; nt<(Nt-N_T/2); nt++) E[nt]=0; 

 fftwt_plan plan = fftwt_plan_dft_1d(Nt, (fftwt_complex*)E, (fftwt_complex*)Er, FFTW_BACKWARD, FFTW_ESTIMATE); 
 fftwt_execute(plan); 
 fftwt_destroy_plan(plan); 

 free(E); 

#ifndef _UNWRAP_FRQUENCIES
 for (int nt=0; nt<Nt; nt++) Er[nt] *= exp(j*OMEGA0*(TMIN+dt*nt*ATOMIC_TIME))*(float_type)FIELD_DENOM/(ATOMIC_FIELD_*N_T);
#else
 for (int nt=0; nt<Nt; nt++) Er[nt] *= (float_type)FIELD_DENOM/(ATOMIC_FIELD_);
#endif

 float_type dx = Xmax/Nx; 

 f_complex* f = (f_complex*)malloc_ch(2*N*Nx*sizeof(f_complex));   for (int n=0; n<2*N; n++) for (int nx=0; nx<Nx; nx++) f[nx+n*Nx]=1.0; 
 
 f_complex* gamma = (f_complex*)malloc_ch(2*N*Nx*sizeof(f_complex));
 f_complex* kappa = (f_complex*)malloc_ch(2*N*Nx*sizeof(f_complex)); 

 float_type* Ixn  = (float_type*)malloc_ch(6*N*sizeof(float_type));


 float_type Eo = real(Er[0]), En = 0; 
 for (int nt=1; nt < Nt; nt++)
 { 
  En = real(Er[nt]);
  for (int n=0; n<2*N; n++) { gamma[n*Nx] = 0.0; kappa[n*Nx] = 1.0; }
  for (int nx=1; nx<(Nx-1); nx++)
  {
   float_type x = (nx+1)*dx; 
   float_type A = dt/dx/dx - dt*(1.0/x - 1.0)/2.0/dx;
   float_type C = dt/dx/dx + dt*(1.0/x - 1.0)/2.0/dx; 
   for (int n=0; n<2*N; n++)
   {	
	float_type z = (n<N) ? Zeff[n] : (-Zeff[n-N]);
        float_type Enc = En/z/z/z;
	float_type Eoc = Eo/z/z/z; 
	f_complex B = f_complex(-2.0*dt/dx/dx + dt*x*Enc/2.0, 1.0); 
    	f_complex D = f_complex(-2.0*dt/dx/dx + dt*x*Eoc/2.0,-1.0); 
    
    	f_complex F = -A*f[nx-1 + n*Nx] - C*f[nx + 1 + n*Nx] - D*f[nx + n*Nx]; 
    
        f_complex G = (A*kappa[nx-1+n*Nx]+B);
        kappa[nx + n*Nx] = -C/G; 
        gamma[nx + n*Nx] = (F-A*gamma[nx -1 + n*Nx])/G;
   } 
  }
  
  for (int n=0; n<2*N; n++) f[Nx-1 + n*Nx] = gamma[Nx-2 + n*Nx]/((float_type)1.0-kappa[Nx-2 + n*Nx]);
  for (int nx=Nx-1; nx>0; nx--) for (int n=0; n<2*N; n++) f[nx-1 + n*Nx] = kappa[nx-1 + n*Nx]*f[nx + n*Nx] + gamma[nx-1 + n*Nx]; 
  Eo = En; 	

  if ((nt % tfac) == 0)
  {
   for (int i=0; i<3; i++) 
   {
 //   float_type x1 = dx, F1 = exp(-x1)*pow(x1,i);
    for (int n=0; n<2*N; n++) Ixn[i+3*n] = 0; 
    for (int nx=0; nx<Nx; nx++)
    {
     float_type x =   (nx+1)*dx, F1 = exp(-x)*pow(x, i);
     for (int n=0; n<2*N; n++) Ixn[i+3*n] += abs2(f[nx+n*Nx])*F1; 
     //F1 = F2; 
    }
    for (int n=0; n<2*N; n++) Ixn[i+3*n] *= dx; 
   }
   NL_SMALLBUFFER1[nt/tfac] = 0; 
   for (int n=0; n<N; n++) NL_SMALLBUFFER1[nt/tfac] += (Ixn[2+3*n]*Ixn[0+3*(n+N)] - Ixn[0+3*n]*Ixn[2+3*(n+N)])/(Ixn[1+3*n]*Ixn[0+3*(n+N)] + Ixn[0+3*n]*Ixn[1+3*(n+N)])*alpha[n]/Zeff[n]; 
   NL_SMALLBUFFER1[nt/tfac] *= 0.5*ELECTRON_CHARGE*BOHR_RADIUS*NEUTRAL_DENSITY;  
  }
 }
	
 free(f); free(gamma); free(kappa); free(Er); free(Ixn); 

#ifndef _UNWRAP_FREQUENCIES 
 NL_SMALLBUFFER1[0] = 0; for (int nt=1; nt<N_T; nt++) NL_SMALLBUFFER1[nt] *= exp(-j*OMEGA0*(TMIN+TSTEP*nt));
#endif 

 fftwt_execute_dft(FFT_FWPLAN_T, (fftwt_complex*)NL_SMALLBUFFER1, (fftwt_complex*)output);
}


void calculate_plasmadensity_2float(f_complex* input, float_type* output, size_t N)
{
//#ifndef _ENABLE_CUDA
 for (size_t i=0; i<N; i++) calculate_plasmadensity_losses_small(input+N_T*i, output+N_T*i);
 return;
/*#else	

 f_complex* cuda_field_piece, *cuda_buf; float_type *cuda_plasma_piece;  

 size_t device_freemem=0, device_totalmem=0;
 int device_processN; MPI_Comm_size(DEVICE_COMM, &device_processN);

 (hipMemGetInfo(&device_freemem, &device_totalmem));

 size_t cuda_pointN = exp2(floor(log2((double)device_freemem/N_T/sizeof(f_complex)/1.5/device_processN)));
 if (N < cuda_pointN) cuda_pointN = N;

 int    blocksize = 64;  if (blocksize > cuda_pointN) blocksize=cuda_pointN;
 size_t cuda_piece_size  = N_T*cuda_pointN*sizeof(f_complex);
 size_t cuda_piece_sizef = N_T*cuda_pointN*sizeof(float_type);

 (hipMalloc((void**)&cuda_field_piece,       cuda_piece_size ));
 (hipMalloc((void**)&cuda_buf,               cuda_piece_size ));
 (hipMalloc((void**)&cuda_plasma_piece,      cuda_piece_sizef));

 for (long i=0; i<N; i+= cuda_pointN)
 {
  if ((N-i)<cuda_pointN) 
  {
	  cuda_pointN = N-i;
	  cuda_piece_size  = N_T*cuda_pointN*sizeof(f_complex);
	  cuda_piece_sizef = N_T*cuda_pointN*sizeof(float_type);
  }

  (hipMemcpy(cuda_field_piece,    input+i*N_T,   cuda_piece_size, hipMemcpyHostToDevice)) ;
  int blockN = cuda_pointN/blocksize, r = cuda_pointN % blocksize;
  calculate_plasma_2float_kernel<<<blockN,blocksize>>>(cuda_field_piece, cuda_plasma_piece, cuda_buf);
  if (r != 0) calculate_plasma_2float_kernel<<<1,r>>>(cuda_field_piece+N_T*blockN*blocksize, cuda_plasma_piece+N_T*blockN*blocksize, cuda_buf+N_T*blockN*blocksize);
  (hipMemcpy(output+i*N_T, cuda_plasma_piece,    cuda_piece_sizef, hipMemcpyDeviceToHost)) ;
 }
  
 (hipFree(cuda_field_piece)); 
 (hipFree(cuda_buf));
 (hipFree(cuda_plasma_piece)); 
#endif*/
}

void calculate_maxplasmadensity_2float(f_complex* input, float_type* output, size_t N)
{
//#ifndef _ENABLE_CUDA

  float_type* b = (float_type*)NL_SMALLBUFFER1;
  for (size_t i=0; i<N; i++) 
  {
	  float_type ro=0;
	  calculate_plasmadensity_losses_small(input+N_T*i, b); 
	  for (size_t nt=0; nt<N_T; nt++) ro=max(ro, b[nt]);
	  output[i]=ro;
  }

/*#else
 f_complex* cuda_field_piece, *cuda_buf; float_type *cuda_maxplasma_piece;  

 size_t device_freemem=0, device_totalmem=0;
 int device_processN; MPI_Comm_size(DEVICE_COMM, &device_processN);

 (hipMemGetInfo(&device_freemem, &device_totalmem));

 size_t cuda_pointN = exp2(floor(log2((double)device_freemem/(N_T+0.5)/sizeof(f_complex)/device_processN))); 
 if (cuda_pointN > N) cuda_pointN=N;

 int    blocksize = 64;  if (blocksize > cuda_pointN) blocksize=cuda_pointN;
 size_t cuda_piece_size  = N_T*cuda_pointN*sizeof(f_complex);
 size_t cuda_piece_sizef =     cuda_pointN*sizeof(float_type);

 (hipMalloc((void**)&cuda_field_piece,       cuda_piece_size ));
 (hipMalloc((void**)&cuda_buf,               cuda_piece_size ));
 (hipMalloc((void**)&cuda_maxplasma_piece,   cuda_piece_sizef));

 for (long i=0; i<N; i+= cuda_pointN)
 {
  if ((N-i)<cuda_pointN) 
  {
	  cuda_pointN = N-i;
	  cuda_piece_size  = N_T*cuda_pointN*sizeof(f_complex);
	  cuda_piece_sizef =     cuda_pointN*sizeof(float_type);
  }

  (hipMemcpy(cuda_field_piece,    input+i*N_T,   cuda_piece_size, hipMemcpyHostToDevice)) ;
  int blockN = cuda_pointN/blocksize, r = cuda_pointN % blocksize;
  calculate_maxplasma_kernel<<<blockN,blocksize>>>(cuda_field_piece, cuda_maxplasma_piece, cuda_buf);
  if (r != 0) calculate_maxplasma_kernel<<<1,r>>>(cuda_field_piece+N_T*blockN*blocksize, cuda_maxplasma_piece+blockN*blocksize, cuda_buf+N_T*blockN*blocksize);
  (hipMemcpy(output+i*N_T, cuda_maxplasma_piece,    cuda_piece_sizef, hipMemcpyDeviceToHost)) ;
 }
  
 (hipFree(cuda_field_piece)); 
 (hipFree(cuda_maxplasma_piece)); 
 (hipFree(cuda_buf));	
#endif*/
}





//----------------------------------------------
//------------------------------------------------


#ifdef _ENABLE_CUDA

void cuda_load_const()
{
 int deviceN = 0; 
 (hipGetDeviceCount(&deviceN)); 
 if (deviceN == 0) throw "No CUDA-enabled devices found.";
 int devID_start = load_namedint(SC_FID, "DEVICE_ID_START", true, 0);
 int mydevice = 0;

#ifndef _GPU_CLUSTER_MAPPING
 if (ISMASTER) printf("\n %d CUDA devices found; process number = %d", deviceN, PROCESS_N); 
 
 if (PROCESS_N <= deviceN)  mydevice = (PROCESS_RANK+devID_start) % deviceN;
 else mydevice = (PROCESS_RANK*deviceN/PROCESS_N + devID_start)%deviceN;
#else
 mydevice = PROCESS_RANK%deviceN;
 if (ISMASTER) printf("\n %d CUDA devices per node", deviceN);
#endif

 (hipSetDevice(mydevice)); 
 MPI_Comm_split(MPI_COMM_WORLD, mydevice, PROCESS_RANK, &DEVICE_COMM);
 int localrank = 0; MPI_Comm_rank(DEVICE_COMM, &localrank);

 printf("\n[%d]:Assigning to device %d, local rank %d.", PROCESS_RANK, mydevice, localrank);


 float_type* plasma_func_device, *raman_func_device;
 float_type* omega_device, *wavenum_device, *hodisp_device;

 float_type pf_re = real(PLASMA_FACTOR); 
 float_type pf_im = imag(PLASMA_FACTOR);

 (hipMalloc(&plasma_func_device, sizeof(f_complex)*N_T)); 
 (hipMalloc(&raman_func_device,  sizeof(f_complex)*N_T));

 (hipMalloc(&wavenum_device,    sizeof(f_complex)*N_T));
 (hipMalloc(&hodisp_device,     sizeof(float_type)*N_T));
 (hipMalloc(&omega_device,      sizeof(float_type)*N_T));

 (hipMemcpy(plasma_func_device, PLASMA_FUNC,        sizeof(f_complex)*N_T, hipMemcpyHostToDevice));
 (hipMemcpy(raman_func_device,  RAMAN_FUNCTION,     sizeof(f_complex)*N_T, hipMemcpyHostToDevice));

 (hipMemcpy(wavenum_device,  WAVENUMBER,     sizeof(f_complex) *N_T, hipMemcpyHostToDevice));
 (hipMemcpy(hodisp_device,   HO_DISPERSION,  sizeof(float_type)*N_T, hipMemcpyHostToDevice));
 (hipMemcpy(omega_device,    OMEGA,          sizeof(float_type)*N_T, hipMemcpyHostToDevice));

 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_plasma_factor_re"),(void*)&pf_re,      sizeof(float_type),0,hipMemcpyHostToDevice));
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_plasma_factor_im"),(void*)&pf_im,      sizeof(float_type),0,hipMemcpyHostToDevice));

 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_plasma_func"),  (void*)&plasma_func_device, sizeof(float_type*),0,hipMemcpyHostToDevice));
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_raman_func"),   (void*)&raman_func_device,  sizeof(float_type*),0,hipMemcpyHostToDevice));
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_wavenum"),      (void*)&wavenum_device,    sizeof(float_type*),0,hipMemcpyHostToDevice));
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_omega"),        (void*)&omega_device,      sizeof(float_type*),0,hipMemcpyHostToDevice));
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_hodisp"),        (void*)&hodisp_device,    sizeof(float_type*),0,hipMemcpyHostToDevice));

  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_N_T"),  (void*)&N_T,  sizeof(int),       0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_TMIN"), (void*)&TMIN,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_TMAX"), (void*)&TMAX,  sizeof(float_type),0,hipMemcpyHostToDevice));
  float_type tstep = (TMAX-TMIN)/N_T; 
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_TSTEP"),(void*)&tstep, sizeof(float_type),0,hipMemcpyHostToDevice)); 

  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_IONIZATION_POTENTIAL"),  (void*)&IONIZATION_POTENTIAL,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_AVALANCHE_CROSSSECTION"), (void*)&AVALANCHE_CROSSSECTION,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_RECOMBINATION_TAU"), (void*)&RECOMBINATION_TAU,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_GROUP_VELOCITY"), (void*)&GROUP_VELOCITY,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_RECOMBINATION_TAU"), (void*)&RECOMBINATION_TAU,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_AMBIENT_CARRIER_DENSITY"), (void*)&AMBIENT_CARRIER_DENSITY,  sizeof(float_type),0,hipMemcpyHostToDevice));

  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_N2"), (void*)&NONLIN_REFRINDEX ,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_N4"), (void*)&NONLIN_REFRINDEX4,  sizeof(float_type),0,hipMemcpyHostToDevice));

  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_PONDEROMOTIVE_COEFFICIENT"), (void*)&PONDEROMOTIVE_COEFFICIENT,  sizeof(float_type),0,hipMemcpyHostToDevice));

#ifdef MULTI_LEVEL_IONIZATION 
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_IONIZATION_LEVEL_N"),     (void*)&IONIZATION_LEVEL_N, sizeof(int), 0, hipMemcpyHostToDevice)); 
  
  float_type* Us_device;  (hipMalloc(&Us_device,    IONIZATION_LEVEL_N*sizeof(float_type))); 
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_IONIZATION_POTENTIALS"),   (void*)&Us_device, sizeof(float_type*), 0, hipMemcpyHostToDevice)); 
  (hipMemcpy(Us_device, IONIZATION_POTENTIALS,      IONIZATION_LEVEL_N*sizeof(float_type), hipMemcpyHostToDevice)); 

  if (N_T < IONIZATION_LEVEL_N*2) throw "cuda_load_const(): N_T should be greater than 2 IONIZATION_LEVEL_N, because one of nonlinear-response buffers is used to store ion densities and ionization rates during plasma density calculation.";
  int ionization_levels_n = IONIZATION_LEVEL_N;
#else 
  int ionization_levels_n  = 1;  
#endif
  float_type* Wln_device; (hipMalloc(&Wln_device, IONIZATION_N*ionization_levels_n*sizeof(float_type))); 
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_IONIZATION_RATE_LN"), (void*)&Wln_device, sizeof(float_type*),0, hipMemcpyHostToDevice));
  (hipMemcpy(Wln_device, IONIZATION_RATE_LN, IONIZATION_N*ionization_levels_n*sizeof(float_type), hipMemcpyHostToDevice));

  float_type kxstep = 2*M_PI/(XMAX-XMIN), kystep = 2*M_PI/(YMAX-YMIN);
 
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_kxstep"), (void*)&kxstep,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_kystep"), (void*)&kystep,  sizeof(float_type),0,hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_OMEGA_MAX"), (void*)&OMEGA_MAX, sizeof(float_type), 0, hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_OMEGA_MIN"), (void*)&OMEGA_MIN, sizeof(float_type), 0, hipMemcpyHostToDevice));
 
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_NEUTRAL_DENSITY"), (void*)&NEUTRAL_DENSITY, sizeof(float_type), 0, hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_RAMAN_FRACTION"), (void*)&RAMAN_FRACTION, sizeof(float_type), 0, hipMemcpyHostToDevice));

  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_wavenum0"), (void*)&WAVENUMBER0, sizeof(float_type), 0, hipMemcpyHostToDevice));
  (hipMemcpyToSymbol(HIP_SYMBOL("cuda_omega0"),   (void*)&OMEGA0,      sizeof(float_type), 0, hipMemcpyHostToDevice));

#ifdef MULTIPHOTON_IONIZATION
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_K_MPI"), (void*)&K_MPI, sizeof(int), 0, hipMemcpyHostToDevice));
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_BETA_MPI_LN"), (void*)&BETA_MPI_LN, sizeof(float_type), 0, hipMemcpyHostToDevice));
#endif

#ifdef THIRD_HARMONICS
 (hipMemcpyToSymbol(HIP_SYMBOL("cuda_th_factor"), (void*)&TH_FACTOR, sizeof(float_type), 0, hipMemcpyHostToDevice)); 
#endif
}


void cuda_free_const()
{
 float_type* pplasma_func, **praman_func, *pwavenum, *pomega, *phodisp;

 (hipMemcpyFromSymbol(&pplasma_func, HIP_SYMBOL("cuda_plasma_func"),  sizeof(float_type*), 0, hipMemcpyDeviceToHost)); hipFree(pplasma_func);
 (hipMemcpyFromSymbol(&praman_func,  HIP_SYMBOL("cuda_raman_func"),   sizeof(float_type*), 0, hipMemcpyDeviceToHost)); hipFree(praman_func);
 (hipMemcpyFromSymbol(&pwavenum,     HIP_SYMBOL("cuda_wavenum"),      sizeof(float_type*), 0, hipMemcpyDeviceToHost)); hipFree(pwavenum);
 (hipMemcpyFromSymbol(&pomega,       HIP_SYMBOL("cuda_omega"),        sizeof(float_type*), 0, hipMemcpyDeviceToHost)); hipFree(pomega);
 (hipMemcpyFromSymbol(&phodisp,      HIP_SYMBOL("cuda_hodisp"),       sizeof(float_type*), 0, hipMemcpyDeviceToHost)); hipFree(phodisp);

 if (cuda_use_pinned_memory)
 {
  hipHostUnregister(FIELD); hipHostUnregister(BIGBUFFER1); hipHostUnregister(BIGBUFFER2);
 }

 if (!cuda_do_extra_memtransfer)
 {
  hipFree(cuda_field); hipFree(cuda_buf1); hipFree(cuda_buf2);
  hipFree(cuda_bufs);
 }
 
 
}


__global__ void calculate_Lresponse_kernel(float_type* field, float_type* buf1, float_type* kt2_, float_type ZSTEP, size_t N_T, size_t n)
{
  size_t  nlocal = blockDim.x*blockIdx.x+threadIdx.x;
 

  for (size_t nw=0; nw<N_T; nw++)
  {
    size_t ofs = 2*(nw + N_T*nlocal); 
#ifndef NO_SPACE_TIME_FOCUSING
     float_type k1  = cuda_wavenum[2*nw];
#else
     float_type k1 = cuda_wavenum0;
#endif	
     float_type k2 = cuda_wavenum[2*nw+1];
     float_type w = cuda_omega[nw]; 
	 float_type kt2n = kt2_[nlocal+n]/k1/k1;
#ifndef NO_DIFFRACTION
    if (w < 0 || kt2n > MAX_KT2) { for (size_t i=0; i<2; i++) field[ofs+i]=0.0; continue;}
#else
    if (w < 0) { for (size_t i=0; i<2; i++) field[ofs+i]=0.0; continue;}
#endif
     
     float_type reE = field[ofs], imE = field[ofs+1];
#ifdef PARABOLICAL_DIFRACTION

	
#ifndef NO_DIFFRACTION
     	float_type phi = ZSTEP*(k1*kt2n/(float_type)2.0 - cuda_hodisp[nw]);
#else
	 	float_type phi = ZSTEP*(-cuda_hodisp[nw]);
#endif
	 float_type M   = exp_f(ZSTEP*k2);
	 float_type c,s;
	 sincos_p(phi,&s,&c);

#ifdef NONLINEARITY_ON      
     reE += ZSTEP*buf1[ofs]; 
     imE += ZSTEP*buf1[ofs+1];
     field[ofs]   = (reE*c - imE*s)*M;// + ZSTEP*buf1[ofs];
     field[ofs+1] = (imE*c + reE*s)*M;// + ZSTEP*buf1[ofs+1];
#endif

#else
	 float_type sqHO = device_sqrtHO(-kt2n); 
	 float_type phi = ZSTEP*(-cuda_hodisp[nw] - k1*sqHO);
	 float_type M   = exp_f(ZSTEP*k2);  
	 float_type c,s;
	 sincos_p(phi,&s,&c);
#ifdef NONLINEARITY_ON      
     reE += ZSTEP*buf1[ofs]  /(1.0+sqHO);   
     imE += ZSTEP*buf1[ofs+1]/(1.0+sqHO); 
     field[ofs]   = (reE*c - imE*s)*M;// + ZSTEP*buf1[ofs]  /(1+sqHO);
     field[ofs+1] = (imE*c + reE*s)*M;// + ZSTEP*buf1[ofs+1]/(1+sqHO);
#endif

#endif


#ifndef NONLINEARITY_ON
     field[ofs]   = (reE*c - imE*s)*M;
     field[ofs+1] = (imE*c + reE*s)*M;
#endif
   }
}


__global__ void calculate_NLresponse_kernel_strided(f_complex* input, f_complex* output, float_type zstep, int N_T, float_type* bufs, float_type* maxA2buf, float_type* maxNL2buf)
{
  
 int ofs =          N_T*(blockDim.x*blockIdx.x   + threadIdx.x);
#ifdef _ODE_RK4
 int bufofs =      20*N_T*(blockDim.x*blockIdx.x)  + threadIdx.x; 
#endif
#ifdef _ODE_HEUN
 int bufofs =      16*N_T*(blockDim.x*blockIdx.x)  + threadIdx.x; 
#endif
 #ifdef _ODE_EULER
 int bufofs =      14*N_T*(blockDim.x*blockIdx.x)  + threadIdx.x; 
#endif


 float_type* inp  = (float_type*)(input +ofs);
 float_type* outp = (float_type*)(output+ofs);


 int stride = blockDim.x;
 int stridei = 31-__clz(stride);
 int ntstride = N_T * 2*stride ;

 float_type* b   = bufs+bufofs;
 float_type* input_buf   = b+2*N_T*stride;
 float_type* output_buf1 = b+4*N_T*stride;

#ifdef _ODE_RK4
 float_type* temp_bufs   = b+12*N_T*stride;
#endif
#ifdef _ODE_HEUN
 float_type* temp_bufs   = b+8*N_T*stride;
#endif
#ifdef _ODE_EULER
 float_type* temp_bufs   = b+6*N_T*stride;
#endif


 float_type maxI_ = 0, maxNL2_ = 0;
 //Runge-Kutta 4-step:
 for (unsigned int i=0; i<2*N_T; i++) 
 {
	 b[i << stridei] = inp[i];
 }

 for (int i=0; i<2*N_T; i++) input_buf[i*stride]=b[i*stride];

 for (int i=0; i<N_T; i++) maxI_ = max_p(maxI_, cuda_abs2(b[ 2*i   *stride], b[(2*i+1)*stride]));

#ifdef _ODE_RK4
 float_type runge_kutta_factor[3] = {0.5,0.5,1};
 for (int nr=0; nr<4; nr++)
 {
    if (nr > 0) for (int i=0; i<2*N_T; i++) input_buf[i*stride] = b[i*stride] + runge_kutta_factor[nr-1]*zstep*output_buf1[i*stride+(nr-1)*ntstride];
	calculate_single_NLresponse_kernel_strided(input_buf, output_buf1+nr*ntstride, N_T, temp_bufs);
 }
#endif 

#ifdef _ODE_HEUN
 for (int nr=0; nr<2; nr++)
 {
    if (nr > 0) for (int i=0; i<2*N_T; i++) input_buf[i*stride] = b[i*stride] + zstep*output_buf1[i*stride+(nr-1)*ntstride];
	calculate_single_NLresponse_kernel_strided(input_buf, output_buf1+nr*ntstride, N_T, temp_bufs);
 }
#endif 

#ifdef _ODE_EULER
 calculate_single_NLresponse_kernel_strided(input_buf, output_buf1, N_T, temp_bufs);
#endif 

 for (unsigned int nt=0; nt<N_T; nt++)
 {
#ifdef _ODE_EULER
  float_type reout = output_buf1[2*nt*stride]; float_type imout=output_buf1[(2*nt+1)*stride]; 
#endif
#ifdef _ODE_HEUN
  float_type reout = (output_buf1[ 2*nt   *stride] + output_buf1[ 2*nt   *stride + ntstride])/2.0; 
  float_type imout = (output_buf1[(2*nt+1)*stride] + output_buf1[(2*nt+1)*stride + ntstride])/2.0; 
#endif
#ifdef _ODE_RK4
  float_type reout = (output_buf1[ 2*nt   *stride] + 2.0*output_buf1[ 2*nt   *stride + ntstride] + 2.0*output_buf1[ 2*nt   *stride + 2*ntstride] + output_buf1[ 2*nt   *stride + 3*ntstride])/6.0; 
  float_type imout = (output_buf1[(2*nt+1)*stride] + 2.0*output_buf1[(2*nt+1)*stride + ntstride] + 2.0*output_buf1[(2*nt+1)*stride + 2*ntstride] + output_buf1[(2*nt+1)*stride + 3*ntstride])/6.0; 
#endif
  maxNL2_ = max_p(maxNL2_, cuda_abs2(reout, imout)); 
  outp[2*nt] = reout; outp[2*nt+1]=imout; 
 }

 maxA2buf[blockIdx.x*blockDim.x + threadIdx.x] = maxI_; 
 maxNL2buf[blockIdx.x*blockDim.x + threadIdx.x] = 2.31; //maxNL2_; 
  
}



__device__ void calculate_single_NLresponse_kernel_strided(float_type* input, float_type* output, int N_T, float_type* bufs)
{
 int stride = blockDim.x;
#ifdef PLASMA_FULL_DISPERSION
 for (int nw=0; nw<2*N_T; nw++) bufs[(2*N_T+nw)*stride] = input[nw*stride];
#endif
 fft_device_strided(input, N_T, 1, stride);
 calculate_plasmadensity_losses_small_device(input, bufs, stride, 2, bufs+4*N_T*stride, bufs+6*N_T*stride); 

 
#ifndef NO_PLASMARESPONSE
#ifndef PLASMA_DISPERSION
#ifndef PLASMA_FULL_DISPERSION 
for (int nt=0; nt<N_T; nt++)
 {
   float_type reE = input[2*nt*stride], imE = input[(2*nt+1)*stride];
   float_type ro  = bufs[(2*nt)*stride];

   float_type re_Fro  = -cuda_plasma_factor_re*ro;
   float_type im_Fro  = -cuda_plasma_factor_im*ro;
   float_type resFro, imsFro;
   if (fabs(re_Fro) < 0.1) device_sqrtHO(re_Fro,im_Fro, &resFro, &imsFro);
   else {sqrtc(1+re_Fro, im_Fro, &resFro, &imsFro);  resFro -= 1;}

   float_type reR  =  cuda_wavenum0* imsFro;
   float_type imR  = -cuda_wavenum0* resFro; 
 
   bufs[(4*N_T+2*nt)  *stride]  += reR*reE - imR*imE;
   bufs[(4*N_T+2*nt+1)*stride]  += reR*imE + imR*reE;
 }
#else
 for (int nt=0; nt<N_T; nt++)
 {
   float_type ro  = bufs[(2*nt)*stride]; 
   float_type reM = (float_type)1.0/(float_type)N_T;
   float_type imM = 0; 
   float_type rew, imw;  sincos_f((float_type)2.0*M_PI*(float_type)nt/(float_type)N_T, &imw, &rew); 
   bufs[(4*N_T+2*nt)*stride] = 0; bufs[(4*N_T+2*nt+1)*stride]=0;

   for (int nw=0; nw<N_T; nw++)
   {
   	float_type reT1, imT1, reT2, imT2;
	reT1  = cuda_plasma_func[2*nw]*ro; imT1  = cuda_plasma_func[2*nw+1]*ro;
 	if (fabs(reT1) < 0.1) device_sqrtHO(reT1,imT1, &reT2, &imT2);
   	else {sqrtc(1+reT1, imT1, &reT2, &imT2);  reT2 -= 1.0;} 		
	reT1 = cuda_wavenum[2*nw]; imT1 = reT1*imT2; reT1 = reT1*reT2;    //T1 = k[w];  T1 = T1*T2 => T1 = T2*k(w

	float_type reAw = bufs[(2*N_T+2*nw)*stride], imAw = bufs[(2*N_T+2*nw+1)*stride];  

	reT2 = reT1*reAw - imT1*imAw; imT2 = reT1*imAw + imT1*reAw; //T2 = T1*A(w)  => T2 = T2*k(w)*A(w)

  	
	bufs[(4*N_T+2*nt)*stride]   +=  (reT2*imM + imT2*reM);      //bufs += -i*k(w)*T2*A(w)*M;  where M = exp(i*2*pi*nt*nw/N_T); 
	bufs[(4*N_T+2*nt+1)*stride] += -(reT2*reM - imT2*imM);

	reT1 = reM; imT1 = imM;		  // M=M*exp(i*2*pi*nt/N_T); 
	reM = reT1*rew - imT1*imw; 
	imM = reT1*imw + imT1*rew; 
   }	
 }
#endif

 fft_device_strided(bufs+4*N_T*stride, N_T, -1, stride);


#else
  for (int nt=0; nt<N_T; nt++)
  {
   float_type reE = input[2*nt*stride], imE = input[(2*nt+1)*stride];
   float_type ro  = bufs[(2*nt)*stride];
   
   bufs[(2*N_T+2*nt)  *stride] = ro*reE;
   bufs[(2*N_T+2*nt+1)*stride] = ro*imE;
  }

 

  fft_device_strided(bufs+4*N_T*stride, N_T, -1,stride); fft_device_strided(bufs+2*N_T*stride, N_T, -1, stride);

  for (int nw=0; nw<N_T; nw++) 
  {
   bufs[(4*N_T+2*nw)  *stride] += cuda_plasma_func[2*nw]*bufs[(2*N_T+2*nw)  *stride] - cuda_plasma_func[2*nw+1]*bufs[(2*N_T+2*nw+1)*stride];
   bufs[(4*N_T+2*nw+1)*stride] += cuda_plasma_func[2*nw]*bufs[(2*N_T+2*nw+1)*stride] + cuda_plasma_func[2*nw+1]*bufs[(2*N_T+2*nw)  *stride];
  }
#endif

#else
 for (int nw=0; nw<2*N_T; nw++) bufs[(4*N_T+nw)  *stride] = 0; 
#endif

 for (int nt=0; nt<N_T; nt++) {bufs[ 2*nt   *stride] = cuda_abs2(input[2*nt*stride], input[(2*nt+1)*stride]); bufs[(2*nt+1)*stride] = 0;}
 fft_device_strided(bufs, N_T, -1, stride);

 for (int nw=0; nw<N_T; nw++)
 {
  float_type reIw = bufs[2*nw*stride], imIw = bufs[(2*nw+1)*stride];
  bufs[(2*N_T+2*nw)  *stride] = cuda_raman_func[2*nw]*reIw - cuda_raman_func[2*nw+1]*imIw; 
  bufs[(2*N_T+2*nw+1)*stride] = cuda_raman_func[2*nw]*imIw + cuda_raman_func[2*nw+1]*reIw;
 }
 fft_device_strided(bufs+2*N_T*stride, N_T,  1, stride); 

 for (int nt=0; nt<N_T; nt++)
 {
  float_type reE  = input[2*nt*stride], imE = input[(2*nt+1)*stride]; 
  float_type I    = cuda_abs2(reE, imE); 
#ifdef THIRD_HARMONICS 
#ifndef _UNWRAP_FREQUENCIES
  float_type carrier_phase = 2.0*cuda_omega0*(cuda_TMIN + nt*cuda_TSTEP);
#else  
  float_type carrier_phase = 0; 
#endif 

  float_type s,c;     sincos_p(carrier_phase,&s,&c);
  float_type s2, c2;  sincos_p(2.0*carrier_phase, &s2, &c2);
  float_type reE2 = reE*reE - imE*imE; 
  float_type imE2 = 2.0*reE*imE;
  float_type reE4 = reE2*reE2 - imE2*imE2;
  float_type imE4 = 2.0*reE2*imE2; 

  float_type reRf = cuda_N2*(bufs[(2*N_T+2*nt)*stride]   + (1-cuda_RAMAN_FRACTION)*cuda_th_factor/3.0*(reE2*c - imE2*s)) + cuda_N4*(I*I + I*(reE2*c - imE2*s)/2.0 + (reE4*c2 - imE4*s2)/10.0);
  float_type imRf = cuda_N2*(bufs[(2*N_T+2*nt+1)*stride] + (1-cuda_RAMAN_FRACTION)*cuda_th_factor/3.0*(reE2*s + imE2*c)) + cuda_N4*(      I*(reE2*s + imE2*c)/2.0 + (reE4*s2 - imE4*c2)/10.0);
#else 
  float_type reRf = cuda_N2*bufs[(2*N_T+2*nt)*stride] + cuda_N4*I*I; 
  float_type imRf = cuda_N2*bufs[(2*N_T+2*nt+1)*stride] + cuda_N4*I*I;
#endif 

  bufs[ 2*nt   *stride] = reRf*reE - imRf*imE; 
  bufs[(2*nt+1)*stride] = imRf*reE + reRf*imE;
 }
 fft_device_strided(bufs, N_T, -1, stride);

 for (int nw=0; nw<N_T; nw++)
 { 
#ifndef NO_SHOCK
  float_type w = cuda_omega[nw];
#else
  float_type w = cuda_omega0;
#endif
  float_type reRf = bufs[2*nw*stride], imRf = bufs[(2*nw+1)*stride]; 
  bufs[(2*N_T+2*nw)  *stride]  =   imRf*w/LIGHT_VELOCITY;
  bufs[(2*N_T+2*nw+1)*stride]  =  -reRf*w/LIGHT_VELOCITY;
 }


#ifdef NONLINEARITY_ON
 for (int i=0; i<2*N_T; i++)
 {
   float_type w = cuda_omega[i>>1]; if (w<0) {output[i*stride]=0.0; continue;} 
   output[i*stride]=(bufs[(4*N_T+i)*stride]+bufs[(2*N_T+i)*stride])*device_hfgaussfilter(w)*device_lfgaussfilter(w);
 }
#else
 for (int i=0; i<2*N_T; i++) output[i*stride]=0.0;
#endif
//for (int i=0; i<2*N_T; i++) output[i*stride]=bufs[(2*N_T+i)*stride];
}



__device__ inline float_type plasma_source_function_device(float_type reA, float_type imA, float_type ro)
{
	return photoionization_function_device(reA, imA, ro)  + avalanche_ionization_function_device(reA, imA, ro) - recombination_function_device(ro);
}


__device__ inline float_type avalanche_ionization_function_device(float_type reA, float_type imA, float_type ro)
{
	if (ro < 0 || ro > cuda_NEUTRAL_DENSITY) return 0;
	float_type I = cuda_abs2(reA, imA); 
    return cuda_AVALANCHE_CROSSSECTION/(cuda_IONIZATION_POTENTIAL+cuda_PONDEROMOTIVE_COEFFICIENT*I)*I*ro*(1-ro/cuda_NEUTRAL_DENSITY);
}

__device__ inline float_type recombination_function_device(float_type ro)
{
	return (ro>0)?(ro/cuda_RECOMBINATION_TAU):0;
}

__device__ inline float_type photoabsorbtion_function_device(float_type reA, float_type imA, float_type ro)
{
	float_type I = cuda_abs2(reA,imA);
	return ((I > IONIZATION_MIN_I/INTENSITY_DENOM)?((float_type)0.5*photoionization_function_device(reA, imA,ro)*((cuda_IONIZATION_POTENTIAL+cuda_PONDEROMOTIVE_COEFFICIENT*I)/I)):(0));
}

__device__ inline void calculate_plasmadensity_small_device(float_type* field, float_type* pro, float_type* buf)
{
        calculate_plasmadensity_small_device_strided(field, pro, 1, buf);
}

__device__ inline void calculate_plasmadensity_small_device_strided(float_type* field, float_type* pro, int stride, float_type* buf)
{
   calculate_plasmadensity_losses_small_device(field, pro, stride, 2, NULL, buf); 
}

__device__  void calculate_plasmadensity_losses_small_device(float_type* field, float_type* pro, int stride, int rostride, float_type* loss, float_type* buf)
{
    float_type tstep = cuda_TSTEP*IONRATE_DENOM;
#ifndef MULTI_LEVEL_IONIZATION
    if (pro) pro[0]=0; 
	if (loss) {loss[0]=0; loss[stride]=0;}
    float_type ro1 = 0;
	float_type ro2 = 0;

	float_type W10, W11, W20, W21, k1, k2;
#else
    float_type* W = buf + cuda_IONIZATION_LEVEL_N*stride; 
    float_type* ro = buf;  for (int i=0; i<cuda_IONIZATION_LEVEL_N; i++) ro[i*stride]=0;
    float_type  tro = 0, tl = 0;
    if (pro) pro[0] = 0; 
    if (loss) {loss[0]=0; loss[stride]=0;}
#endif 

    float_type reE0 = field[0],        imE0 = field[stride];
    float_type reE1=0, imE1=0, I0=cuda_abs2(reE0, imE0), I1=0;
      
   for (int nt=1; nt<cuda_N_T;   nt++)
   {
	  // Plasma density calculation in performed via Runge-Khutta method of second order (Heun method)
	  reE1 = field[(2*nt  )*stride];
	  imE1 = field[(2*nt+1)*stride];
	  I1 = cuda_abs2(reE1, imE1);
//	  printf("\nE1=%g+i%g, nt=%d, stride=%d, I1 = %g",reE1, imE1, nt, stride, I1);
#ifndef MULTI_LEVEL_IONIZATION
	  W10 = photoionization_function_device(reE0, imE0, ro1);    k1 = (tstep)*(W10 + avalanche_ionization_function_device(reE0, imE0, ro1) - recombination_function_device(ro1));	    
	  W11 = photoionization_function_device(reE1, imE1, ro1+k1); k2 = (tstep)*(W11 + avalanche_ionization_function_device(reE1, imE1, ro1+k1) - recombination_function_device(ro1+k1));	    
	  ro1 += 0.5*(k1+k2); if (ro1 > cuda_NEUTRAL_DENSITY) ro1 = cuda_NEUTRAL_DENSITY;

	  
	  if (pro)  pro[rostride*nt*stride] = ro1;
	  k1 = (I1>IONIZATION_MIN_I)?(-0.5*(W10*(cuda_IONIZATION_POTENTIAL+cuda_PONDEROMOTIVE_COEFFICIENT*I0)/I0)):0;
 
	  if (loss) {loss[(2*nt-2)*stride] = reE1*k1; loss[(2*nt-1)*stride]=imE1*k1;}
#else
	 if (loss) tl=0; 
	 float_type k1=0, k2=0;
	 if (I0 > IONIZATION_MIN_I/INTENSITY_DENOM)
	 {

	   photoionization_functionsN_device(reE0, imE0,     W,                                stride);
	   photoionization_functionsN_device(reE1, imE1,     W+cuda_IONIZATION_LEVEL_N*stride, stride);
	   k1 = tstep*W[0]*(cuda_NEUTRAL_DENSITY-ro[0]); k2 = tstep*W[cuda_IONIZATION_LEVEL_N*stride]*(cuda_NEUTRAL_DENSITY-ro[0]-k1); 
	   if (k1 < 0) k1=0; if (k2 < 0) k2 = 0;
	   ro[0] += 0.5*(k1+k2); if (ro[0] > cuda_NEUTRAL_DENSITY) ro[0]=cuda_NEUTRAL_DENSITY; 
	   tro = ro[0]; 
	
           tl = -(float_type)0.5*k1/tstep*(cuda_IONIZATION_POTENTIALS[0]+cuda_PONDEROMOTIVE_COEFFICIENT*I0)/I0;
	 
	   for (int n=1; n<cuda_IONIZATION_LEVEL_N; n++) 
	   {
 	      k1 = tstep*W[n*stride]*(ro[(n-1)*stride]-ro[n*stride]); k2 = tstep*W[(n+cuda_IONIZATION_LEVEL_N)*stride]*(ro[(n-1)*stride]-ro[n*stride]-k1); 
	      if (k1 < 0) k1=0; if (k2 < 0) k2 = 0;
	      ro[n*stride] = ro[n*stride] + 0.5*(k1+k2); 
	      if (ro[n*stride]>ro[(n-1)*stride]) ro[n*stride]=ro[(n-1)*stride];
	      tro  += ro[n*stride];
              tl -= (float_type)0.5*k1/tstep*(cuda_IONIZATION_POTENTIALS[n]+cuda_PONDEROMOTIVE_COEFFICIENT*I0)/I0;
	   }
	  }
  	  if (pro) {pro[rostride*stride*nt]=tro;}
          if (loss) {loss[(2*nt-2)*stride] = tl*reE0; loss[(2*nt-1)*stride]=tl*imE0;}
#endif
	  reE0=reE1; imE0 = imE1;
	  I0 = I1;
   } 
   if (loss) {loss[(2*cuda_N_T-2)*stride]=0; loss[(2*cuda_N_T-1)*stride]=0;}
}

__device__ inline void calculate_plasmadensity_small_device_strided_2float(float_type* field, float_type* pro, int stride, float_type* buf)
{
   calculate_plasmadensity_losses_small_device(field, pro, stride, 1, NULL, buf);
}

__device__ inline float_type calculate_maxplasmadensity_small_device(float_type* field, float_type* buf)
{
 //calculate_plasmadensity_losses_small_device(field, NULL, blockDim.x*gridDim.x, 1, NULL, buf);
 return 0;//buf[cuda_N_T-1];
}


__device__ float_type photoionization_function_device(float_type reA, float_type imA, float_type ro)
{
        float_type I = cuda_abs2(reA, imA); 
		if (ro > cuda_NEUTRAL_DENSITY) return 0;
        if (I > (IONIZATION_MIN_I/INTENSITY_DENOM))
	{
	     float_type lnI = log_p(I)+INTENSITY_DENOM_LN;
#ifdef MULTIPHOTON_IONIZATION
         return 0;
	//return ((ro<cuda_NEUTRAL_DENSITY)?(exp_p(cuda_K_MPI*lnI+cuda_BETA_MPI_LN-IONRATE_DENOM_LN)*(cuda_NEUTRAL_DENSITY-ro)):0);
#else

         int n = (float_type)(floor((lnI - IONIZATION_MIN_I_LN)/IONIZATION_I_LN_TOLERANCE));
         if (n < 0 || n > (IONIZATION_N-2)) return 0; 
         float_type lnIn  = IONIZATION_MIN_I_LN+n*IONIZATION_I_LN_TOLERANCE;

         float_type lnWn  = cuda_IONIZATION_RATE_LN[n];
         float_type lnWnp = cuda_IONIZATION_RATE_LN[n+1];
#ifdef IONIZATION_LINEAR_INTERP
#ifdef IONIZATION_GAS
		 float_type Wn = exp_p(lnWn+log_p(cuda_NEUTRAL_DENSITY-ro)), Wnp = exp(lnWnp+log_p(cuda_NEUTRAL_DENSITY-ro));
#else
		 float_type Wn = exp_p(lnWn), Wnp = exp_p(lnWnp);
#endif
		 float_type In = exp_p(lnIn), Inp = exp_p(lnIn+IONIZATION_I_LN_TOLERANCE);
		 return Wn + (I-In)*(Wnp-Wn)/(Inp-In);
#else
         float_type lnW = lnWn + (lnI-lnIn)*(lnWnp-lnWn)/IONIZATION_I_LN_TOLERANCE;

#ifdef IONIZATION_GAS
		 return exp_p(lnW + log_p(cuda_NEUTRAL_DENSITY-ro));
#else
         return exp_p(lnW); 
#endif
#endif
	}
	return 0;
#endif
}


#ifdef MULTI_LEVEL_IONIZATION

__device__ void photoionization_functionsN_device(float_type reA, float_type imA, float_type* W, int stride)
{

        float_type I = cuda_abs2(reA, imA);
        if (I < (IONIZATION_MIN_I/INTENSITY_DENOM)) {for (int i=0; i<cuda_IONIZATION_LEVEL_N; i++) W[i*stride]=0; return;}

	float_type lnI = log(I)+INTENSITY_DENOM_LN;
         int n = (int)floor((lnI - IONIZATION_MIN_I_LN)/IONIZATION_I_LN_TOLERANCE);
        if (n > IONIZATION_N-2 || n < 0) {for (int i=0; i<cuda_IONIZATION_LEVEL_N; i++) W[i*stride]=0; return;}  
	
#ifdef YUDIN_IVANOV_CORRECTION
         float_type theta = atan(imA/reA);  
         float_type wl = cuda_omega0*PLANCK_CONSTANT_REDUCED/HARTREE_ENERGY; 
	 float_type phase_v = cuda_omega0/cuda_wavenum0;
         float_type E = exp(lnI/2)*sqrt(2*VACUUM_PERMEABILITY*phase_v*phase_v/cuda_GROUP_VELOCITY);
         float_type T  = (E*E/ATOMIC_FIELD/ATOMIC_FIELD)/wl/wl/wl;
#endif
       
	for (int i=0; i<cuda_IONIZATION_LEVEL_N; i++)
	{
         float_type lnIn  = IONIZATION_MIN_I_LN+n*IONIZATION_I_LN_TOLERANCE;
 	 float_type lnWn  = cuda_IONIZATION_RATE_LN[IONIZATION_N*(i)+n];
         float_type lnWnp = cuda_IONIZATION_RATE_LN[IONIZATION_N*(i)+n+1];
	 float_type lnW = lnWn + (lnI-lnIn)*(lnWnp-lnWn)/IONIZATION_I_LN_TOLERANCE;

	 float_type F = 1.0; 
  #ifdef PPT_IONIZATION
  #ifdef YUDIN_IVANOV_CORRECTION
	float_type g = sqrt(2.0*cuda_IONIZATION_POTENTIALS[i]*INTENSITY_DENOM/IONRATE_DENOM/ELECTRON_CHARGE/ELECTRON_CHARGE*ELECTRON_MASS)*cuda_omega0/E;

	if (g < MAX_YI_GAMMA) F = exp(-T*(YI_Phi_device(theta, g)-YI_Phi_device(0,g)));         
  #endif
  #endif

	 W[i*stride] = exp(lnW)*F; 
	}
}
#endif


__device__  float_type photoionization_function_device2(float_type reA, float_type imA, float_type ro1, float_type ro2)
{
        float_type I = cuda_abs2(reA, imA); 
		if (ro2 > ro1) return 0;
		if (ro1 < 1e-5*cuda_NEUTRAL_DENSITY) return 0;
        if (I > (IONIZATION_MIN_I/INTENSITY_DENOM))
	{
	     float_type lnI = log_p(I)+INTENSITY_DENOM_LN;

         int n = (float_type)(floor((lnI - IONIZATION_MIN_I_LN)/IONIZATION_I_LN_TOLERANCE));
         if (n < 0 || n > (IONIZATION_N-2)) return 0; 
         float_type lnIn  = IONIZATION_MIN_I_LN+n*IONIZATION_I_LN_TOLERANCE;

         float_type lnWn  = cuda_IONIZATION_RATE_LN[IONIZATION_N+n];
         float_type lnWnp = cuda_IONIZATION_RATE_LN[IONIZATION_N+n+1];

		  float_type lnW = lnWn + (lnI-lnIn)*(lnWnp-lnWn)/IONIZATION_I_LN_TOLERANCE;

#ifdef IONIZATION_GAS
		 return exp_p(lnW + log_p(ro1-ro2));
#else
         return exp_p(lnW); 
#endif
	}
	return 0;
}




__global__ void calculate_plasma_2float_kernel(f_complex* input, float_type* output, f_complex* buf)
{
	int ofs = cuda_N_T*(blockDim.x*blockIdx.x   + threadIdx.x);
	calculate_plasmadensity_small_device_strided_2float((float_type*)(input+ofs), output+ofs, 1, (float_type*)(buf+ofs));
}


__global__ void calculate_maxplasma_kernel   (f_complex* input, float_type* output, f_complex* buf)
{
	int ofs = blockDim.x*blockIdx.x   + threadIdx.x;
	float_type maxro_ = calculate_maxplasmadensity_small_device((float_type*)(input+cuda_N_T*ofs), (float_type*)(buf+cuda_N_T*ofs)); 
	output[ofs] = maxro_;
}


#ifdef YUDIN_IVANOV_CORRECTION

__device__ float_type YI_Phi_device(float_type theta, float_type g)
{
 float_type sth = fabs(sin(theta)); 
 float_type sth2 = sth*sth; 

 float_type a = 1.0 + g*g - sth2;
 float_type b = sqrt(a*a + 4*g*g*sth2); 
 float_type c = sqrt(pow(sqrt((a+b)/2.0)+g, 2.0) + pow(sqrt((b-a)/2.0)+sth, 2.0)); 
 
 return ((g*g + sth2 + 0.5)*log(c) - (3.0*sqrt((b-a)/2.0)/2.0)*sth - sqrt((a+b)/2.0)/2.0*g); 
}

#endif



#endif 
