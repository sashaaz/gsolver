
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void cudaadd(float* cA, float* cB, float* cC);

const int N = 32;
int main()
{
 int deviceN = 0; //Number of CUDA-enabled GPUs (graphics cards)
 hipGetDeviceCount(&deviceN); 
 if (deviceN == 0) {printf("Error! No cuda-enabled devices found!"); return 1;}
 hipSetDevice(0);  //Set 0-th device as active. Previous versions of CUDA didn't alow use of multiple GPU in a single program (except mutlti-thread programs)
// The latest CUDA release has included this featurem but I've never used it, and don't know how it works.

 float* A = (float*)malloc(N*sizeof(float));  //GeForce GPU's (like we have here) are supposed to work much faster with single-precision floating point numbers (float type) rather then 
//with double type.
 float* B = (float*)malloc(N*sizeof(float));
 float* C = (float*)malloc(N*sizeof(float));

 for (int i=0; i<N; i++) {A[i]=i%5 + i/100.0;  B[i]=2.0 + i;}  //fill in arrays

 float* cA = NULL; 
 float* cB = NULL;
 float* cC = NULL;

 hipMalloc(&cA, N*sizeof(float)); //Allocate memory in GPU. 
 hipMalloc(&cB, N*sizeof(float));
 hipMalloc(&cC, N*sizeof(float));

 hipMemcpy(cA, A, N*sizeof(float), hipMemcpyHostToDevice); //copy arrays A and B to GPU. 
 hipMemcpy(cB, B, N*sizeof(float), hipMemcpyHostToDevice);
// Take care! cA and cB point to address in GPU memory. You cannot directly write there (e.g. like cA[i] = 10.1). You MUST use cudaMemcpy

 cudaadd<<<1,N>>>(cA, cB, cC);  //call GPU procedure (or "kernel"), using 1 block with N threads in block

 hipMemcpy(C, cC, N*sizeof(float), hipMemcpyDeviceToHost);

 for (int i=0; i<N; i++) printf("\nA[%d]=%g,  B[%d]=%g, C[%d]=%g, should be %g", i, A[i], i, B[i], i, C[i], A[i]+B[i]);

 free(A); free(B); free(C);                      //Free host arrays
 hipFree(cA); hipFree(cB); hipFree(cC);       //Free GPU arrays
 return 0;
}


__global__ void cudaadd(float* cA, float* cB, float* cC)
{
 int n = threadIdx.x;
 cC[n] = cA[n]+cB[n];
}
